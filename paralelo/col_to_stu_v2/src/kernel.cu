#include "hip/hip_runtime.h"
#include <kernel.cuh>

__constant__ int d_cupoArray[85];
__constant__ double d_alpha[3];



__global__ void newSolution_kernel(
    double *d_array_current_Solution,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col,
    const int n_students,
    const int n_colegios,
    const int n_thread,
    const double max_dist,
    const int* __restrict__ d_alumnosSep,
    const int totalVuln,
    const int* __restrict__ d_aluxcol,
    const int* __restrict__ d_aluVulxCol,
    const int* __restrict__ d_currentSolution,
    const double* __restrict__ d_distMat,
    const int* __restrict__ d_shuffle_students,
    const int* __restrict__ d_shuffle_colegios,
    const double* __restrict__ d_currentVars,
    size_t pitch){

    /// Shared Memory
    extern __shared__ double sharedMem[];
    double* solutions =(double*)sharedMem;
    int* solutions_col = (int*)&solutions[blockDim.x/32+1];
    int* solutions_alu =  (int*)&solutions_col[blockDim.x/32+1];
    /// Inicializa variables en 0
    int aluchange,
            colchange,
            newSchool,
            aluVulCol= 0,
            aluNoVulCol= 0,
            totalAluCol= 0,
            myID = threadIdx.x,
            currentSchool,
            myCurrentAluxCol = 0,
            myCurrentAluVulxCol = 0,
            myNewAluxCol = 0,
            myNewAluVulxCol = 0;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            var1,
            var2,
            var3,
            sumDist = 0.0;
    /// Inicializa arrays
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    aluchange = d_shuffle_students[tid%n_students]; 
    //aluchange = d_shuffle_students[threadIdx.x]; 
    colchange = d_shuffle_colegios[blockIdx.x%n_colegios];
    currentSchool = d_currentSolution[aluchange];
    //printf("%d|%d|%d|%d\n",colchange,currentSchool,aluchange,tid%n_students);

    double cost_solution = 9999.9;
    int col_solution = colchange;
    int alu_solution = aluchange;


    newSchool = colchange;
    sumDist= d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];
    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    sumDist-=cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + currentSchool]);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual 
    totalcostCupo-=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela nueva
    totalcostCupo-=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2)));
    ////////////////////////////////////////////////////////////////
    /////// Realiza Movimiento
    ////////////////////////////////////////////////////////////////
    //ELimina el estudiante de la escuela actual
    myCurrentAluxCol = d_aluxcol[currentSchool]-1;
    myCurrentAluVulxCol = d_aluVulxCol[currentSchool]-d_alumnosSep[aluchange];
    //Asigna al estudiante a la nueva escuela
    myNewAluxCol = d_aluxcol[newSchool]+1;
    myNewAluVulxCol = d_aluVulxCol[newSchool]+d_alumnosSep[aluchange];

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist+=cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + newSchool]);
    // seg de la escuela actual
    totalAluCol = myCurrentAluxCol;
    aluVulCol = myCurrentAluVulxCol;
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual
    totalcostCupo+=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    
    // seg de la escuela antigua
    totalAluCol = myNewAluxCol;
    aluVulCol = myNewAluVulxCol;
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela antigua
    totalcostCupo+=cu_round_n(((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2))));


    var1 = (sumDist/n_students);
    var1= (var1/max_dist);
    //cout << var1 << "\n";
    var2 = (totalSesc/2.0);
    //cout << var2 << "\n";
    var3 = (totalcostCupo /n_colegios);
    cost_solution =  (double)((d_alpha[0]*var1)+(d_alpha[1]*var2)+(d_alpha[2]*var3));
    //printf("%.16lf %d %d\n",solutions[myID], colchange,aluchange);
    __syncthreads();

    int warpID = threadIdx.x / 32;
    int lane = threadIdx.x % 32;
    #define FULL_MASK 0xffffffff
    // Encuentra minimo a nivel de warp
    for(int salto=32/2; salto>0; salto>>=1){ // salto>>=1 es igual a salto/2 
        double neighbour_solution = __shfl_down_sync(FULL_MASK,cost_solution,salto);
        int co = __shfl_down_sync(FULL_MASK,col_solution,salto);
        int al = __shfl_down_sync(FULL_MASK,alu_solution,salto);
        if(neighbour_solution < cost_solution){
            cost_solution = neighbour_solution;
            col_solution = co;
            alu_solution = al;
        }
        __syncthreads();
    }
    if(lane==0){
        solutions[warpID] = cost_solution;
        solutions_col[warpID] = col_solution;
        solutions_alu[warpID] = alu_solution;
    }
    
    __syncthreads();
    // Encuentra el minimo a nivel de bloque
    if(warpID == 0){
        cost_solution = (myID < blockDim.x/32)?solutions[lane]:9999.9;
        col_solution = (myID < blockDim.x/32)?solutions_col[lane]:0;
        alu_solution = (myID < blockDim.x/32)?solutions_alu[lane]:0;
        //printf("laneID= %d %.16lf %d %d\n",lane, cost_solution, alu_solution, col_solution);
        for(int salto=32/2; salto >0; salto>>=1){
            double neighbour_solution = __shfl_down_sync(FULL_MASK,cost_solution,salto);
            int co = __shfl_down_sync(FULL_MASK,col_solution,salto);
            int al = __shfl_down_sync(FULL_MASK,alu_solution,salto);
            if(neighbour_solution < cost_solution){
                cost_solution = neighbour_solution;
                col_solution = co;
                alu_solution = al;
            }
        }
        //__syncthreads();
        if(lane==0){
            d_array_current_Solution[blockIdx.x] = cost_solution;
            d_array_current_Solution_alu[blockIdx.x] = alu_solution;
            d_array_current_Solution_col[blockIdx.x] = col_solution;
            //printf("laneID= %d %.16lf %d %d\n",lane, cost_solution, alu_solution, col_solution);
        }
       
    }

    
}

__global__ void reduce_block_kernel(
    double *d_array_current_Solution,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col){

    extern __shared__ double sharedMem[];
    double* solutions =(double*)sharedMem;
    int* solutions_col = (int*)&solutions[blockDim.x/32+1];
    int* solutions_alu =  (int*)&solutions_col[blockDim.x/32+1];
    int myID = threadIdx.x;
    int end = blockDim.x-1;


    double cost_solution = d_array_current_Solution[myID];
    int col_solution = d_array_current_Solution_col[myID];
    int alu_solution = d_array_current_Solution_alu[myID];
    int warpID = threadIdx.x / 32;
    int lane = threadIdx.x % 32;
    #define FULL_MASK 0xffffffff
    
    if(myID==0){
        if(d_array_current_Solution[end] < cost_solution){
            cost_solution = d_array_current_Solution[end];
            col_solution = d_array_current_Solution_col[end];
            alu_solution = d_array_current_Solution_alu[end];
        }
    }




    // Encuentra minimo a nivel de warp
    //printf("%.16lf %d %d\n", cost_solution,col_solution,alu_solution);
    for(int salto=32/2; salto>0; salto>>=1){ // salto>>=1 es igual a salto/2 
        double neighbour_solution = __shfl_down_sync(FULL_MASK,cost_solution,salto);
        int co = __shfl_down_sync(FULL_MASK,col_solution,salto);
        int al = __shfl_down_sync(FULL_MASK,alu_solution,salto);
        if(neighbour_solution < cost_solution){
            cost_solution = neighbour_solution;
            col_solution = co;
            alu_solution = al;
        }
    }
    if(lane==0){
        solutions[warpID] = cost_solution;
        solutions_col[warpID] = col_solution;
        solutions_alu[warpID] = alu_solution;
    }
    


    __syncthreads();
    // Encuentra el minimo a nivel de bloque
    if(warpID == 0){
        cost_solution = (myID < blockDim.x/32)?solutions[lane]:9999.9;
        col_solution = (myID < blockDim.x/32)?solutions_col[lane]:0;
        alu_solution = (myID < blockDim.x/32)?solutions_alu[lane]:0;
        //printf("laneID= %d %.16lf %d %d\n",lane, cost_solution, alu_solution, col_solution);
        for(int salto=32/2; salto >0; salto>>=1){
            double neighbour_solution = __shfl_down_sync(FULL_MASK,cost_solution,salto);
            int co = __shfl_down_sync(FULL_MASK,col_solution,salto);
            int al = __shfl_down_sync(FULL_MASK,alu_solution,salto);
            if(neighbour_solution < cost_solution){
                cost_solution = neighbour_solution;
                col_solution = co;
                alu_solution = al;
            }
        }
        //__syncthreads();
        if(lane==0){
            d_array_current_Solution[blockIdx.x] = cost_solution;
            d_array_current_Solution_alu[blockIdx.x] = alu_solution;
            d_array_current_Solution_col[blockIdx.x] = col_solution;
            //printf("laneID= %d %.16lf %d %d\n",lane, cost_solution, alu_solution, col_solution);
        }
       
    }
    /*
    __syncthreads();
    while(salto){
        if(salto-(myID+1)>myID){
            if(solutions[salto-(myID+1)]<solutions[myID]){
                solutions[myID]=solutions[salto-(myID+1)];
                solutions_alu[myID]=solutions_alu[salto-(myID+1)];
                solutions_col[myID]=solutions_col[salto-(myID+1)];
            }
        }
        salto = (salto/2)+(salto&(2-1));
        if(salto==1){
            salto = 0;
        }
        __syncthreads();
    }
    if(myID==0)
    {
        //printf("\n %.10lf\n ",solutions[myID]);
        d_array_current_Solution[0] = solutions[0];
        d_array_current_Solution_alu[0] = solutions_alu[0];
        d_array_current_Solution_col[0] = solutions_col[0];
        //printf("%d \t %.20lf | %d %d \n",blockIdx.x,solutions[0],solutions_alu[0],solutions_col[0]);
    }
    */
}

__global__ void calculateSolution(
    double *d_array_current_Solution,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col,
    const int n_students,
    const int n_colegios,
    const int n_thread,
    const double max_dist,
    const int* __restrict__ d_alumnosSep,
    int totalVuln,
    int* d_aluxcol,
    int* d_aluVulxCol,
    int* d_currentSolution,
    const double* __restrict__ d_distMat,
    size_t pitch,
    double *d_currentVars,
    double *d_costCurrentSolution){

    int aluchange,
    colchange,
    newSchool,
    aluVulCol= 0,
    aluNoVulCol= 0,
    totalAluCol= 0,
    currentSchool;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            var1,
            var2,
            var3,
            sumDist = 0.0;
    /// Inicializa arrays

    aluchange = d_array_current_Solution_alu[0];
    colchange = d_array_current_Solution_col[0];
    currentSchool = d_currentSolution[aluchange];
    //printf("%d \t %.20lf | %d %d %d \n",blockIdx.x,d_array_current_Solution[0],d_array_current_Solution_alu[0],d_array_current_Solution_col[0],currentSchool);
    newSchool = colchange;

    
    sumDist= d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];
    //printf("%lf |%lf |%lf |%lf |%d |%d \n",sumDist,totalSesc,totalcostCupo,d_array_current_Solution[0],aluchange,colchange);

    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    sumDist-=cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + currentSchool]);
    //printf("%lf \n",sumDist);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual 
    //printf("%lf \n",totalSesc);
    totalcostCupo-=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    //printf("%lf \n",totalcostCupo);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    
    totalSesc-=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela nueva

    totalcostCupo-=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2)));
    //printf("a%d \n",newSchool);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    ////////////////////////////////////////////////////////////////
    /////// Realiza Movimiento
    ////////////////////////////////////////////////////////////////
    //ELimina el estudiante de la escuela actual
    d_aluxcol[currentSchool]-=1;
    d_aluVulxCol[currentSchool]-=d_alumnosSep[aluchange];
    //Asigna al estudiante a la nueva escuela
    d_currentSolution[aluchange] = newSchool;
    d_aluxcol[newSchool]+=1;
    d_aluVulxCol[newSchool]+=d_alumnosSep[aluchange];

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist+=cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + newSchool]);
    
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual

    totalcostCupo+=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    //printf("%lf \n",totalcostCupo);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    // seg de la escuela antigua
    totalAluCol = d_aluxcol[newSchool];

    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela antigua

    totalcostCupo+=cu_round_n(((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2))));
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    d_currentVars[0] = sumDist;
    d_currentVars[1] = totalSesc;
    d_currentVars[2] = totalcostCupo;
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    var1 = (sumDist/n_students);
    var1= (var1/max_dist);
    //cout << var1 << "\n";
    var2 = (totalSesc/2.0);
    //cout << var2 << "\n";
    var3 = (totalcostCupo /n_colegios);
    d_costCurrentSolution[0] =  (double)((d_alpha[0]*var1)+(d_alpha[1]*var2)+(d_alpha[2]*var3));
    d_array_current_Solution[0] = d_costCurrentSolution[0];
    if(d_array_current_Solution[0]!=d_costCurrentSolution[0]){
        printf("ERRORRRRRRRRRRRR no son iguales!!!!!!!!!!!!!!!!!!\n");
        printf("%.10lf\n",d_array_current_Solution[0]);
        printf("%.10lf\n",d_costCurrentSolution[0]);

    }
    //d_array_current_Solution[0] = d_costCurrentSolution[0];
}


__global__ void copyMemSolution(
    int *solution,
    int *new_solution,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        solution[i] = new_solution[i];
    }
}
__global__ void copyMemCol(
    int *col,
    int *new_col,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        col[i] = new_col[i];
    }
}
__global__ void copyVars(
    double *var,
    double *new_var){

    var[threadIdx.x] = new_var[threadIdx.x];
}

__global__ void copyCost(
    double *costCurrentSolution,
    double *new_costCurrentSolution
    ){

        costCurrentSolution[0] = new_costCurrentSolution[0];

    }

inline __device__ double cu_round_n(double x)
{
    double digits = pow(10.0, 16);
    return trunc(x * digits) / digits;
}
