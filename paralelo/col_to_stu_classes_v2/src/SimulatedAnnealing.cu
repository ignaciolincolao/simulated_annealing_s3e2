#include "hip/hip_runtime.h"
#include <SimulatedAnnealing.cuh>
#include <CUDAWrapper.cuh>

#include <limits>

#ifndef SAVE_DATA
#define SAVE_DATA 0
#endif
#define DECIMAL 16

typedef std::numeric_limits<double> dbl;


SimulatedAnnealing::SimulatedAnnealing(AcceptanceCriterion* AC,
    CoolingScheme* CS,
    LengthTemperature* LT,
    ReheatingMethod* RM,
    Dataset* DS,
    RecordManager* RMgr,
    SimulatedParams* saParams,
    CUDAParams* cuParams,
    mt19937& mt)
    : 
    acceptanceCriterion(AC), 
    coolingScheme(CS),
    lengthTemperature(LT),
    reheatingMethod(RM),
    dataSet(DS),
    recordManager(RMgr),
    saParams(*saParams),
    cuParams(*cuParams),
    acParams(AC->getAcParams()),
    csParams(CS->getCsParams()),
    ltParams(LT->getLtParams()),
    rmParams(RM->getRmParams()),
    rmgrParams(RMgr->getRmgrParams()),
    mt(mt), 
    dist(0, 0), 
    dist2(0, 0), 
    dist_accepta(0.0, 1.0)
    {      
        mt.seed(saParams->seed);
    }
    
/*
static std::mutex addInfoMutex;

static void addInfoToSave(RecordManager *recordManager,
                          double costCurrentSolution,
                          double meanDist,
                          double S,
                          double costCupo,
                          SimulatedParams *saParams)
{
    std::lock_guard<std::mutex> lock(addInfoMutex);
    recordManager->vector_costCurrentSolution.push_back(costCurrentSolution);
    recordManager->vector_meanDist.push_back(meanDist);
    recordManager->vector_segregation.push_back(S);
    recordManager->vector_costoCupo.push_back(costCupo);
    recordManager->vector_temp.push_back(saParams->temp);
    recordManager->vector_count.push_back(saParams->count);
}
*/
double SimulatedAnnealing::runGPU(){
    CUDAWrapper* cudaWrapper = new CUDAWrapper(cuParams, saParams, mt);
    // cout << "test" << endl;
    inicializationValues(cudaWrapper);
    cudaWrapper->memInit(previousSolution,
        bestSolution,
        currentSolution,
        cupoArray,
        alumnosSep,
        totalVuln,
        aluxcol,
        aluVulxCol,
        matrestest,
        alpha,
        currentVars);

    cout << "--------------- Primeros datos -------------\n";
    cout << "Primer costo de solución: " << costBestSolution << "\n";
    cout << "Primer distancia: " << meanDist(currentSolution, distMat) << "\n";
    cout << "Primer Segregación: " << S(currentSolution, alumnosSep, totalVuln) << "\n";
    cout << "Primer CostoCupo: " << costCupo(currentSolution, cupoArray) << "\n\n";
#if SAVE_DATA
    recordManager->openRecordInfo();
    recordManager->openRecordGraphics();
    recordManager->openRecordGraphicsBestSolution();

    recordManager->SaveInfoInit(costBestSolution,
                                meanDist(currentSolution, distMat),
                                S(currentSolution, alumnosSep, totalVuln),
                                costCupo(currentSolution, cupoArray));

    recordManager->SaveGraphicsInit(meanDist(currentSolution, distMat),
                                    S(currentSolution, alumnosSep, totalVuln),
                                    costCupo(currentSolution, cupoArray),
                                    costCurrentSolution);

    recordManager->SaveGraphicsBestSolution(currentSolution);


    recordManager->closeRecordInfo();
    recordManager->closeRecordGraphics();
    recordManager->closeRecordGraphicsBestSolution();
#endif
    ///////////////////////////////////////////////////
    /// Inicio el contador de tiempo antes de iniciar el algortimo
    ///////////////////////////////////////////////////
    auto start = std::chrono::high_resolution_clock::now();
    ///////////////////////////////////////////////////
    /// Comienza a ejecutarse el algoritmo de SA
    ///////////////////////////////////////////////////
    while(saParams.temp > saParams.min_temp){
        ///////////////////////////////////////////////////
        /// Copia Solución Anterior a la actual
        ///////////////////////////////////////////////////
        cudaWrapper->memCopyPrevToCurrent();
        ///////////////////////////////////////////////////
        ///  Selecciona aleatoria mente a los alumnos
        ///////////////////////////////////////////////////
        shuffle(saParams.shuffle_student, saParams.max_changes_students, dist);
        shuffle(saParams.shuffle_colegios, saParams.max_changes_school, dist2);
        ///////////////////////////////////////////////////
        ///  Envia datos a GPU
        ///////////////////////////////////////////////////
        cudaWrapper->uploadCurrentMemorySolution();
        ///////////////////////////////////////////////////
        ///  Ejecuta los kernel
        //////////////////////////////////////////////////
        cudaWrapper->newSolution();
        

    
        ///////////////////////////////////////////////////
        ///  Metodo Nuevo
        //////////////////////////////////////////////////
        cudaWrapper->getCurrentSolutionGpuToHost(costCurrentSolution);  
        if(costCurrentSolution >= costPreviousSolution){
            if(acceptanceCriterionApply() == 1){
                cudaWrapper->newSolutionRandomSelection(dist,
                    dist2);
            }
        }
        ///////////////////////////////////////////////////
        ///  Actualiza la nueva solución en la GPU
        //////////////////////////////////////////////////
        cudaWrapper->newSolutionUpdate(costCurrentSolution);
            
        ///////////////////////////////////////////////////
        ///  Verifica Error
        //////////////////////////////////////////////////
        if(costCurrentSolution<0.00 || isnan(costCurrentSolution)){
            cout << "error" << endl;
            cout << saParams.count << endl;
            std::cout << saParams.shuffle_colegios[cuParams.selectThread] << "\n";
            std::cout << saParams.shuffle_student[cuParams.selectBlock] << "\n";
            std::cout << "distancia: " << meanDist(currentSolution,distMat) << "\n";
            std::cout << "Segregación: " << S(currentSolution,alumnosSep, totalVuln) << "\n";
            std::cout << "CostoCupo: " << costCupo(currentSolution,cupoArray) << "\n";
            std::cout << costCurrentSolution;
            exit(1);
        }
        
        
        ///////////////////////////////////////////////////
        /// 
        //////////////////////////////////////////////////
        if(costCurrentSolution < costBestSolution){
            cudaWrapper->AcceptanceBestSolution();
            costBestSolution = costCurrentSolution;
            costPreviousSolution = costCurrentSolution;
            saParams.c_accepta++;
            saParams.count_rechaso = 0;

            // futures.push_back(std::async(std::launch::async,
            //                   addInfoToSave,
            //                   recordManager,
            //                   costCurrentSolution,
            //                   meanDist(currentSolution, distMat),
            //                   S(currentSolution, alumnosSep, totalVuln),
            //                   costCupo(currentSolution, cupoArray),
            //                   &saParams
            //                 ));
#if SAVE_DATA
            cudaWrapper->copySolutionToHost(bestSolution, previousSolution);
            recordManager->vector_costCurrentSolution.emplace_back(costBestSolution);
            recordManager->vector_meanDist.emplace_back(meanDist(bestSolution, distMat));
            recordManager->vector_segregation.emplace_back(S(bestSolution, alumnosSep, totalVuln));
            recordManager->vector_costoCupo.emplace_back(costCupo(bestSolution, cupoArray));
            recordManager->vector_temp.emplace_back(saParams.temp);
            recordManager->vector_count.emplace_back(saParams.count);
#endif
        }
        else {
            if(acceptanceCriterion->apply(costPreviousSolution,costCurrentSolution,dist_accepta ) == 1) {

                cudaWrapper->AcceptanceSolution();
                costPreviousSolution = costCurrentSolution;

                saParams.count_rechaso = 0;
                saParams.c_accepta++;
            }
            else {
                saParams.count_rechaso++;
                
            }
        }

        if(lengthTemperature->apply()){
            coolingScheme->apply();
        }
        reheatingMethod->apply();
        

        
        //cout << costCurrentSolution << costPreviousSolution << "| |" << saParams.temp << "| |" << saParams.count<< endl;
        cudaWrapper->synchronizeBucle();
        saParams.count_trials++;
        saParams.count++;
    }
    ///////////////////////////////////////////////////
    /// Obtiene el tiempo de ejecución
    ///////////////////////////////////////////////////
    auto end = std::chrono::high_resolution_clock::now();
    double time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_taken *= 1e-9;
    cudaWrapper->copySolutionToHost(bestSolution, previousSolution);

    cout << "--------------- Resultado Final ----------------" << "\n";
    cout << "Numero de Ciclos: " << saParams.count << "\n";
    cout << "Costo de la solución previa: " << costPreviousSolution << "\n";
    cout << "Costo de la mejor solución: " << costBestSolution << "\n";
    cout << "Costo de la solución actual: " << costCurrentSolution << "\n";
    cout << "Tiempo de ejecución de SA: " << time_taken << "\n";
    cout << "distancia: " << meanDist(bestSolution, distMat) << "\n";
    cout << "Segregación: " << S(bestSolution, alumnosSep, totalVuln) << "\n";
    cout << "CostoCupo: " << costCupo(bestSolution, cupoArray) << "\n";
    cout << "--------------- Finalizo con exito ----------------" << "\n";

#if SAVE_DATA
    recordManager->openRecordInfo();
    recordManager->openRecordGraphics();
    recordManager->openRecordGraphicsBestSolution();
    recordManager->openRecordRegister();

    recordManager->SaveInfoFinish(costPreviousSolution,
                                  costBestSolution,
                                  costCurrentSolution,
                                  time_taken,
                                  meanDist(bestSolution, distMat),
                                  S(bestSolution, alumnosSep, totalVuln),
                                  costCupo(bestSolution, cupoArray));

    recordManager->SaveGraphicsFinish();

    recordManager->SaveGraphicsBestSolution(bestSolution);
    recordManager->SaveInfoRegister(
        time_taken,
        costBestSolution,
        meanDist(bestSolution, distMat),
        S(bestSolution, alumnosSep, totalVuln),
        costCupo(bestSolution, cupoArray),
        csParams.coolingRate,
        rmParams.k_reheating_init,
        rmParams.e_const,
        rmParams.k_reheating,
        ltParams.len1_init,
        ltParams.len2_init,
        ltParams.len3_init,
        ltParams.len4_init,
        ltParams.len1,
        ltParams.len2,
        ltParams.len3,
        ltParams.len4,
        acParams.Th,
        cuParams.n_block,
        cuParams.n_thread
    );

    recordManager->closeRecordInfo();
    recordManager->closeRecordGraphics();
    recordManager->closeRecordGraphicsBestSolution();
    recordManager->closeRecordRegister();
#endif

    // cout << "finalizo con :" << costBestSolution << endl;
    return (costBestSolution);
}

template <typename T>
void SimulatedAnnealing::inicializationValues(T* wrapper){
    int x = 0, z = 0;
    totalVuln = dataSet->totalVuln;
    saParams.n_colegios = dataSet->n_colegios;
    saParams.n_students = dataSet->n_students;
    //cout << fixed << setprecision(70) << endl;
    //srand(time(NULL));


    ///////////////////////////////////////////////////
    /// Inicializa Variables y arreglos
    ///////////////////////////////////////////////////

    aluxcol= (int *)malloc(sizeof(int)*saParams.n_colegios);
    aluVulxCol = (int *)malloc(sizeof(int)*saParams.n_colegios);
    previousAluxCol = (int *)malloc(sizeof(int)*saParams.n_colegios);
    previousAluVulxCol = (int *)malloc(sizeof(int)*saParams.n_colegios);
    bestAluxCol = (int *)malloc(sizeof(int)*saParams.n_colegios);
    bestAluVulxCol = (int *)malloc(sizeof(int)*saParams.n_colegios);
    alpha = saParams.alpha;
    ptr_alpha = &saParams.alpha[0];
    
    saParams.count = 0;


    distMat=(double **)malloc(sizeof(double)*saParams.n_students);
    for(x=0; x < saParams.n_students; x++) {
        distMat[ x ]=(double *)malloc(sizeof(double)*saParams.n_colegios);
    }

    wrapper->mallocHost(
        previousSolution,
        bestSolution,
        currentSolution,
        cupoArray,
        alumnosSep,
        matrestest,
        currentVars,
        previousVars,
        bestVars);


    ///////////////////////////////////////////////////
    /// Se asignan las escuelas un arreglo que y estudiantes a la escuela
    /// las escuelas tendran como identificación el indice
    /// y currentSolution tiene como indice al estudiante y el valor del indice a la escuela que asignada
    ///////////////////////////////////////////////////
    initializeArray(aluxcol, 
                previousAluxCol, 
                bestAluxCol, 
                aluVulxCol, 
                previousAluVulxCol, 
                bestAluVulxCol, 
                alumnosSep,
                dataSet->students,
                dataSet->colegios);
    assignSchoolToArray(previousSolution, bestSolution, currentSolution, dataSet->ptr_colegios, dataSet->ptr_students, cupoArray);
    calcDist(dataSet->ptr_colegios, dataSet->ptr_students, distMat);
    saParams.max_dist = getMaxDistance(distMat);
    normalizedAlpha(alpha);

    ///////////////////////////////////////////////////
    /// Registro de datos
    ///////////////////////////////////////////////////
    costBestSolution = calCosto(currentSolution,distMat,ptr_alpha, alumnosSep, totalVuln, cupoArray);
    costPreviousSolution = costBestSolution;
    costCurrentSolution = costBestSolution;

    saParams.count++;

    ///////////////////////////////////////////////////
    /// Genera distribuciones para seleccionar un estudiante y una escuela al azar
    ///////////////////////////////////////////////////

    dist = uniform_int_distribution<int>(0, saParams.n_students-1);
    dist2 = uniform_int_distribution<int>(0, saParams.n_colegios-1);

    saParams.count_rechaso=0;
    saParams.c_accepta = 0;
    saParams.c_cooling_temperature = 0;
    rmParams.k_reheating_init = rmParams.k_reheating;
    saParams.temp_init = saParams.temp;
    saParams.count_trials = 0;
    ltParams.len1_init = ltParams.len1;
    ltParams.len2_init = ltParams.len2;
    ltParams.len3_init = ltParams.len3;
    ltParams.len4_init = ltParams.len4;

    ////////////////////////////////////////////////////////////////////////
    // VARIABLES DE PRUEBA
    ////////////////////////////////////////////////////////////////////////

    
    // double costCurrentSolutionV2 = costCurrentSolution;
    
    currentVars[0] = sumDist(currentSolution,distMat);
    currentVars[1] = sumS(currentSolution, alumnosSep, totalVuln);
    currentVars[2] = sumCostCupo(currentSolution,cupoArray);
    previousVars[0] = currentVars[0];
    previousVars[1] = currentVars[1];
    previousVars[2] = currentVars[2];
    
    double var1,var2,var3;
    var1 = (currentVars[0]/saParams.n_students);
    var1= (var1/saParams.max_dist);
    //cout << var1 << "\n";
    var2 = (currentVars[1]/2.0);
    //cout << var2 << "\n";
    var3 = (currentVars[2] /saParams.n_colegios);
    costBestSolution = (double)((ptr_alpha[0]*var1)+(ptr_alpha[1]*var2)+(ptr_alpha[2]*var3));
    costPreviousSolution = costBestSolution;
    costCurrentSolution = costBestSolution;
    auto start_compare = std::chrono::high_resolution_clock::now();
    auto end_compare = std::chrono::high_resolution_clock::now();
    double time_taken_v1 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_compare - start_compare).count();
    //double time_taken_v2;
    //double vector_time1 =0;
    // double vector_time2=0;
    saParams.max_changes_students = min(cuParams.n_thread*cuParams.n_block, saParams.n_students);
    saParams.max_changes_school = min(cuParams.n_block, saParams.n_colegios);

    double *array_costCurrentSolution = (double *) malloc(sizeof(double) * cuParams.n_block * cuParams.n_thread);
    for (x = 0; x < cuParams.n_block; x++){
        for (z = 0; z < cuParams.n_thread; z++){
            array_costCurrentSolution[cuParams.n_thread * x + z] = 0.0;
        }
    }
    ///////////////////////////////////////////////////
    /// Inicializa las distribuciónes
    ///////////////////////////////////////////////////
    for (int x = 0; x < saParams.n_students; x++) {
        for (int z = 0; z < saParams.n_colegios; z++) {
            matrestest[saParams.n_colegios * x + z] = distMat[x][z];
        }
    }

    dist = std::uniform_int_distribution<int>(0, saParams.n_students-1);
    dist2 = std::uniform_int_distribution<int>(0, saParams.n_colegios-1);
}

///////////////////////////////////////////////////
///////////////////////////////////////////////////


///////////////////////////////////////////////////
/// Calcula el costo
///////////////////////////////////////////////////
double SimulatedAnnealing::calCosto(int *currentSolution, double **distMat, const double *ptr_alpha, int *alumnosSep, int totalVuln, int *cupoArray){
    double var1 = meanDist(currentSolution,distMat)/saParams.max_dist;
    //cout << "distancia: " << var1 << "\n";
    double var2 = S(currentSolution, alumnosSep, totalVuln);
    //cout << "Segregación: " << var2 << "\n";
    double var3 = costCupo(currentSolution,cupoArray);
    //cout << "CostoCupo: " << var3 << "\n";
    return (double)((ptr_alpha[0]*var1)+(ptr_alpha[1]*var2)+(ptr_alpha[2]*var3));
}

///////////////////////////////////////////////////
/// Distancia promedio que recorreran los estudiantes
///////////////////////////////////////////////////
double SimulatedAnnealing::meanDist(const int *currentSolution, double  **distMat){
    double sumDist=0.0;
    for(int i=0;i<saParams.n_students;i++){
        sumDist+=round_n(distMat[i][currentSolution[i]]); // distMat[estudiante][escuela]
    }
    //cout << "meanDist: " << sumDist << endl;
    //cout << "Numero de estudiantes: " << saParams.n_students << "  |  Suma de distancias:" << sumDist << "\n";
    return sumDist/saParams.n_students;
}

double SimulatedAnnealing::sumDist(const int *currentSolution, double  **distMat){
    double sumDist=0.0;
    for(int i=0;i<saParams.n_students;i++){
        sumDist+=round_n(distMat[i][currentSolution[i]]); // distMat[estudiante][escuela]
    }
    //cout << "sumDist: " << sumDist << endl;
    //cout << "Numero de estudiantes: " << saParams.n_students << "  |  Suma de distancias:" << sumDist << "\n";
    return sumDist;
}


///////////////////////////////////////////////////
/// Calcula segregación por duncan
///////////////////////////////////////////////////

double SimulatedAnnealing::S(const int *currentSolution,const int *alumnosSep, int totalVuln){
    double totalSesc = 0.0;
    int aluVulCol =0;
    int aluNoVulCol = 0;
    for(int n=0; n<saParams.n_colegios;n++){
        aluVulCol = 0;
        aluNoVulCol = 0;
        for (int a = 0; a < saParams.n_students; a++){
            if(currentSolution[a] == n){
                aluNoVulCol++;
                aluVulCol+=alumnosSep[a];
            }
        }
        if(aluNoVulCol>0){
            aluNoVulCol =aluNoVulCol - aluVulCol;
            totalSesc+=round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(saParams.n_students-totalVuln))));
        }
    }
    return totalSesc/2.0;
}

double SimulatedAnnealing::sumS(const int *currentSolution,const int *alumnosSep, int totalVuln){
    double totalSesc = 0.0;
    int aluVulCol =0;
    int aluNoVulCol = 0;
    for(int n=0; n<saParams.n_colegios;n++){
        aluVulCol = 0;
        aluNoVulCol = 0;
        for (int a = 0; a < saParams.n_students; a++){
            if(currentSolution[a] == n){
                aluNoVulCol++;
                aluVulCol+=alumnosSep[a];
            }
        }
        if(aluNoVulCol>0){
            aluNoVulCol =aluNoVulCol - aluVulCol;
            totalSesc+=round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(saParams.n_students-totalVuln))));
        }
    }
    return totalSesc;
}


///////////////////////////////////////////////////
/// Calcula el costo de tener los estudiantes en las escuelas
///////////////////////////////////////////////////

double SimulatedAnnealing::costCupo(int *currentSolution,int *cupoArray){
    double totalcostCupo = 0.0;
    int totalAluCol = 0;
    // double a = 0.0;
    for(int j=0;j<saParams.n_colegios;j++){
        totalAluCol = 0;
        for(int i=0; i<saParams.n_students; i++){
            if(currentSolution[i]==j){
                totalAluCol++;
            }
        }
        totalcostCupo+=round_n((double)totalAluCol*fabs(((double)cupoArray[j]-totalAluCol)/pow(((double)cupoArray[j]/2),2)));
    }
    return totalcostCupo/saParams.n_colegios;
}



double SimulatedAnnealing::sumCostCupo(int* currentSolution,int *cupoArray){
    double totalcostCupo = 0.0;
    int totalAluCol = 0;
    for(int j=0;j<saParams.n_colegios;j++){
        totalAluCol = 0;
        for(int i=0; i<saParams.n_students; i++){
            if(currentSolution[i]==j){
                totalAluCol++;
            }
        }
        totalcostCupo+= round_n((double)totalAluCol*fabs(((double)cupoArray[j]-totalAluCol)/pow(((double)cupoArray[j]/2),2)));
    }
    return totalcostCupo;
}
///////////////////////////////////////////////////
/// Genera una nueva solución en donde asigna a un estudiante a una escuela
/// aleatoriamente
///////////////////////////////////////////////////

void SimulatedAnnealing::newSolution(int *currentSolution,const int *previousSolution){
    //random_device rd;
    //mt19937 mt(rd());
    uniform_int_distribution<int> dist(0, saParams.n_students);
    random_device rd2;
    mt19937 mt2(rd2());
    uniform_int_distribution<int> dist2(0, saParams.n_colegios);
    int selectStudent=dist(mt);
    int selectSchool = dist2(mt2);
    for(int x=0; x<saParams.n_students; x++){
        if(x == selectStudent) {
            currentSolution[x] = selectSchool;
        }
        else {
            currentSolution[x] = previousSolution[x];
        }
    }

}




///////////////////////////////////////////////////
/// Asigna a las soluciones la escuela actual Solo se utiliza al inicio
///////////////////////////////////////////////////
void SimulatedAnnealing::assignSchoolToArray(int *previousSolution, int *bestSolution, int *currentSolution, Info_colegio *ptr_colegios, Info_alu *ptr_students, int *cupoArray){
    Info_alu *ptr_aux = ptr_students;
    for(int x=0;x < saParams.n_colegios;x++){
        for(int y=0; y < saParams.n_students; y++){
            if(ptr_colegios->rbd == ptr_students->rbd){
                previousSolution[y] = x;
                bestSolution[y] = x;
                currentSolution[y] = x;
            }
            ptr_students++;

        }
        /*
         * cupoArray sera un arreglo que por indice es la escuela y su valor sera el cupo que posee esa escuela
         * se asume que las escuelas pueden tener sobre cupo.
         */

        cupoArray[x] = ptr_colegios->num_alu+ ((int)((ptr_colegios->num_alu*10)/100));
        ptr_students = ptr_aux;
        ptr_colegios++;
    }
}
///////////////////////////////////////////////////
/// Crea una matriz de distancia donde x es el estudiante, y es la escuela
///////////////////////////////////////////////////
void SimulatedAnnealing::calcDist(Info_colegio *ptr_colegios, Info_alu *ptr_students, double **distMat){
    Info_colegio *ptr_aux = ptr_colegios;
    for(int x=0;x < saParams.n_students ;x++){
        for(int y=0; y < saParams.n_colegios; y++){
            distMat[x][y] = sqrt( pow((ptr_students->latitude - ptr_colegios->latitude),2)+pow((ptr_students->longitude - ptr_colegios->longitude),2))/1000;
            ptr_colegios++;

        }
        ptr_colegios = ptr_aux;
        ptr_students++;
    }
}



void SimulatedAnnealing::shuffle(int *values, const int max_change, uniform_int_distribution<int> distri) {
    int randvalue1,randvalue2,tem_value;
    for (int i = 0; i<max_change; i++) {
        randvalue1 = distri(mt);
        randvalue2 = i;
        tem_value = values[randvalue1];
        values[randvalue1] = values[randvalue2];
        values[randvalue2] = tem_value;
    }
}

////////////////////////////////////////////////
////// Obtiene la maxima distancia que un estudiante podria llegar a recorrer
///////////////////////////////////////////////////
double SimulatedAnnealing::getMaxDistance(double **distMat){
    double max = 0;
    for(int i=0;i<saParams.n_students;i++){
        for(int x=0;x<saParams.n_colegios;x++){
            if(distMat[i][x]>max){
                max = distMat[i][x];
            }
        }
    }
    return max;
}

///////////////////////////////////////////////////
/// Calcula el valor de los alpha
///////////////////////////////////////////////////
void SimulatedAnnealing::normalizedAlpha(double *alpha)
{
    double sumaAlpha = 0.0;
    for(int x=0; x<3; x++){
        sumaAlpha +=alpha[x];
    }
    for(int x=0; x<3; x++){
        alpha[x]= alpha[x]/(double)sumaAlpha;
    }
}


///////////////////////////////////////////////////
/// Asigna Información de las escuelas a best, previus y current soluciones
///////////////////////////////////////////////////
void SimulatedAnnealing::initializeArray(int *aluxcol, int *previousAluxCol, int *bestAluxCol, int *aluVulxCol, int *previousAluVulxCol, int *bestAluVulxCol, int *alumnosSep, vector<Info_alu> &students,vector<Info_colegio> &colegios)
{
    for(int x = 0; x < saParams.n_colegios; x++){
        aluxcol[x] = colegios[x].num_alu;
        previousAluxCol[x] = colegios[x].num_alu;
        bestAluxCol[x] = colegios[x].num_alu;
        aluVulxCol[x] = colegios[x].prioritario;
        previousAluVulxCol[x] = colegios[x].prioritario;
        bestAluVulxCol[x] = colegios[x].prioritario;

    }
    ///////////////////////////////////////////////////
    /// Se crear un arreglo donde el el valor es la posición del estudiante sep
    ///////////////////////////////////////////////////
    for(int x=0; x < saParams.n_students; x++) {
        alumnosSep[x] = students[x].sep;
    }
}


double SimulatedAnnealing::round_n(double x)
{
    double digits = pow(10.0, DECIMAL);
    return trunc(x * digits) / digits;
}


int SimulatedAnnealing::acceptanceCriterionApply() {
    return acceptanceCriterion->apply(costPreviousSolution,costCurrentSolution,dist_accepta);
}


SimulatedAnnealing::~SimulatedAnnealing() {

}

