#include <CUDAWrapper.cuh>
#include <kernel.cuh>

#include <assert.h>
#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

CUDAWrapper::CUDAWrapper(CUDAParams &cuParams, SimulatedParams &saParams, mt19937 &mt)
    : cuParams(cuParams), saParams(saParams), mt(mt) {

    hipDeviceProp_t deviceProp;
    hipGetDevice(&deviceId);

    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId); // Calcula el numero de SMstream
    hipGetDeviceProperties(&deviceProp, 0);
    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;
    nWarp = deviceProp.warpSize;
    streams = new hipStream_t[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamCreate(&streams[i]);
    }
    hipEventCreate(&start_cuda);
    hipEventCreate(&stop_cuda);
}
CUDAWrapper::~CUDAWrapper() {

    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamDestroy(streams[i]);
    }
    hipFree(d_array_current_Solution);
    hipFree(d_costCurrentSolution);
    hipFree(d_costBestSolution);
    hipFree(d_costPreviousSolution);
    hipFree(d_currentVars);
    hipFree(d_bestVars);
    hipFree(d_previousVars);
    hipFree(d_array_current_Solution_alu);
    hipFree(d_array_current_Solution_col);
    hipFree(d_shuffle_colegios);
    hipFree(d_shuffle_students);
    hipFree(d_aluxcol);
    hipFree(d_previousAluxcol);
    hipFree(d_aluVulxCol);
    hipFree(d_previousAluVulxCol);
    hipFree(d_currentSolution);
    hipFree(d_bestSolution);
    hipFree(d_previousSolution);
    hipFree(d_alumnosSep);
    hipFree(d_cupoArray);
    hipFree(d_distMat);
    hipFree(d_alpha);
    hipFree(d_choices);
    hipFree(d_penalty);
    hipEventDestroy(start_cuda);
    hipEventDestroy(stop_cuda);
}

void CUDAWrapper::memInit(
    int *&previousSolution,
    int *&bestSolution,
    int *&currentSolution,
    int *&cupoArray,
    int *&alumnosSep,
    int &totalVuln,
    int *&aluxcol,
    int *&aluVulxCol,
    double *&matrestest,
    double *&alpha,
    uint8_t *&choices,
    double *&currentVars) {
    hipMalloc((void **)&d_array_current_Solution, cuParams.n_block * sizeof(double));
    hipMalloc((void **)&d_costCurrentSolution, 1 * sizeof(double));
    hipMalloc((void **)&d_costBestSolution, 1 * sizeof(double));
    hipMalloc((void **)&d_costPreviousSolution, 1 * sizeof(double));
    hipMalloc((void **)&d_currentVars, 4 * sizeof(double));
    hipMalloc((void **)&d_bestVars, 4 * sizeof(double));
    hipMalloc((void **)&d_previousVars, 4 * sizeof(double));
    hipMalloc((void **)&d_array_current_Solution_alu, cuParams.n_block * sizeof(int));
    hipMalloc((void **)&d_array_current_Solution_col, cuParams.n_block * sizeof(int));
    hipMalloc((void **)&d_shuffle_colegios, saParams.max_changes_school * sizeof(int));
    hipMalloc((void **)&d_shuffle_students, saParams.max_changes_students * sizeof(int));
    hipMalloc((void **)&d_aluxcol, saParams.n_colegios * sizeof(int));
    hipMalloc((void **)&d_previousAluxcol, saParams.n_colegios * sizeof(int));
    hipMalloc((void **)&d_aluVulxCol, saParams.n_colegios * sizeof(int));
    hipMalloc((void **)&d_previousAluVulxCol, saParams.n_colegios * sizeof(int));
    hipMalloc((void **)&d_currentSolution, saParams.n_students * sizeof(int)); // Solución actual
    hipMalloc((void **)&d_bestSolution, saParams.n_students * sizeof(int));
    hipMalloc((void **)&d_previousSolution, saParams.n_students * sizeof(int));
    hipMalloc((void **)&d_alumnosSep, saParams.n_students * sizeof(int)); // arreglo que contiene la id de cada usuario vulnerable
    hipMalloc((void **)&d_cupoArray, saParams.n_colegios * sizeof(int));
    hipMalloc((void **)&d_choices, saParams.n_students * 5 * sizeof(uint8_t));

    ///////////////////////////////////////////////////
    /// Genera arreglos que contendran valores del 0 hasta saParams.n_students y saParams.n_colegios
    ///////////////////////////////////////////////////

    for (int i = 0; i < saParams.n_students; i++) {
        saParams.shuffle_student[i] = i;
    }

    for (int i = 0; i < saParams.n_colegios; i++) {
        saParams.shuffle_colegios[i] = i;
    }

    hipMallocPitch(&d_distMat,
                    &pitch,
                    saParams.n_colegios * sizeof(double),
                    saParams.n_students); // Reserva memoria para la matriz de distancia

    gpuErrchk(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_alpha), alpha, 4 * sizeof(double), 0, hipMemcpyHostToDevice, streams[2]));
    gpuErrchk(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_n_students), &saParams.n_students, sizeof(int), 0, hipMemcpyHostToDevice, streams[3]));
    gpuErrchk(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_n_colegios), &saParams.n_colegios, sizeof(int), 0, hipMemcpyHostToDevice, streams[4]));
    gpuErrchk(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_max_dist), &saParams.max_dist, sizeof(double), 0, hipMemcpyHostToDevice, streams[5]));
    gpuErrchk(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_totalVuln), &totalVuln, sizeof(int), 0, hipMemcpyHostToDevice, streams[6]));
    // gpuErrchk(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_choices), &choices, saParams.n_students * 5, 0, hipMemcpyHostToDevice, streams[9]));

    size_t h_pitchBytes = saParams.n_colegios * sizeof(double);
    hipMemcpy2DAsync(d_distMat,
                      pitch,
                      matrestest,
                      h_pitchBytes,
                      saParams.n_colegios * sizeof(double),
                      saParams.n_students,
                      hipMemcpyHostToDevice,
                      streams[3]);

    hipMemcpyAsync(d_choices, choices, saParams.n_students * 5, hipMemcpyHostToDevice, streams[7]);
    hipMemcpyAsync(d_currentSolution, currentSolution, saParams.n_students * sizeof(int), hipMemcpyHostToDevice, streams[2]);
    hipMemcpyAsync(d_previousSolution, currentSolution, saParams.n_students * sizeof(int), hipMemcpyHostToDevice, streams[3]);
    hipMemcpyAsync(d_bestSolution, currentSolution, saParams.n_students * sizeof(int), hipMemcpyHostToDevice, streams[4]);
    hipMemcpyAsync(d_aluxcol, aluxcol, saParams.n_colegios * sizeof(int), hipMemcpyHostToDevice, streams[5]);
    hipMemcpyAsync(d_previousAluxcol, aluxcol, saParams.n_colegios * sizeof(int), hipMemcpyHostToDevice, streams[6]);
    hipMemcpyAsync(d_aluVulxCol, aluVulxCol, saParams.n_colegios * sizeof(int), hipMemcpyHostToDevice, streams[7]);
    hipMemcpyAsync(d_previousAluVulxCol, aluVulxCol, saParams.n_colegios * sizeof(int), hipMemcpyHostToDevice, streams[8]);
    hipMemcpyAsync(d_currentVars, currentVars, 4 * sizeof(double), hipMemcpyHostToDevice, streams[9]);
    hipMemcpyAsync(d_previousVars, currentVars, 4 * sizeof(double), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_bestVars, currentVars, 4 * sizeof(double), hipMemcpyHostToDevice, streams[1]);
    hipMemcpyAsync(d_alumnosSep, alumnosSep, saParams.n_students * sizeof(int), hipMemcpyHostToDevice, streams[2]);
    hipMemcpyAsync(d_cupoArray, cupoArray, saParams.n_colegios * sizeof(int), hipMemcpyHostToDevice, streams[3]);

    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();

    if (errSync != hipSuccess)
        printf("0 Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("0 Async kernel error: %s\n", hipGetErrorString(errAsync));
}

void CUDAWrapper::memCopyPrevToCurrent() {
    copyMemSolution<<<numberOfBlocks, threadsPerBlock, 0, streams[0]>>>(d_currentSolution, d_previousSolution, saParams.n_students);
    copyMemCol<<<numberOfBlocks, threadsPerBlock, 0, streams[1]>>>(d_aluxcol, d_previousAluxcol, saParams.n_colegios);
    copyMemCol<<<numberOfBlocks, threadsPerBlock, 0, streams[2]>>>(d_aluVulxCol, d_previousAluVulxCol, saParams.n_colegios);
    copyVars<<<1, 4, 0, streams[3]>>>(d_currentVars, d_previousVars);
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("1 Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("1 Async kernel error: %s\n", hipGetErrorString(errAsync));
}

void CUDAWrapper::uploadCurrentMemorySolution() {
    hipMemcpyAsync(d_shuffle_students, saParams.shuffle_student, saParams.max_changes_students * sizeof(int), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_shuffle_colegios, saParams.shuffle_colegios, saParams.max_changes_school * sizeof(int), hipMemcpyHostToDevice, streams[1]);
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("2 Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("2 Async kernel error: %s\n", hipGetErrorString(errAsync));
};

void CUDAWrapper::AcceptanceBestSolution() {
    copyMemSolution<<<numberOfBlocks, threadsPerBlock, 0, streams[0]>>>(d_bestSolution, d_currentSolution, saParams.n_students);
    copyMemSolution<<<numberOfBlocks, threadsPerBlock, 0, streams[1]>>>(d_previousSolution, d_currentSolution, saParams.n_students);
    copyMemCol<<<numberOfBlocks, threadsPerBlock, 0, streams[2]>>>(d_previousAluxcol, d_aluxcol, saParams.n_colegios);
    copyMemCol<<<numberOfBlocks, threadsPerBlock, 0, streams[3]>>>(d_previousAluVulxCol, d_aluVulxCol, saParams.n_colegios);
    copyVars<<<1, 4, 0, streams[4]>>>(d_previousVars, d_currentVars);
    copyVars<<<1, 4, 0, streams[5]>>>(d_bestVars, d_currentVars);
    copyCost<<<1, 1, 0, streams[6]>>>(d_costBestSolution, d_costCurrentSolution);
    copyCost<<<1, 1, 0, streams[7]>>>(d_costPreviousSolution, d_costCurrentSolution);
    // for (int i = 0; i < NUM_STREAMS; ++i) { hipStreamSynchronize(streams[i]); }
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("9 Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("9 Async kernel error: %s\n", hipGetErrorString(errAsync));
}

void CUDAWrapper::AcceptanceSolution() {
    copyMemSolution<<<numberOfBlocks, threadsPerBlock, 0, streams[0]>>>(d_previousSolution, d_currentSolution, saParams.n_students);
    copyMemCol<<<numberOfBlocks, threadsPerBlock, 0, streams[1]>>>(d_previousAluxcol, d_aluxcol, saParams.n_colegios);
    copyMemCol<<<numberOfBlocks, threadsPerBlock, 0, streams[2]>>>(d_previousAluVulxCol, d_aluVulxCol, saParams.n_colegios);
    copyVars<<<1, 4, 0, streams[3]>>>(d_previousVars, d_currentVars);
    copyCost<<<1, 1, 0, streams[4]>>>(d_costPreviousSolution, d_costCurrentSolution);
    // for (int i = 0; i < NUM_STREAMS; ++i) { hipStreamSynchronize(streams[i]); }
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("10 Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("10 Async kernel error: %s\n", hipGetErrorString(errAsync));
}

void CUDAWrapper::newSolution() {
    newSolution_kernel<<<cuParams.n_block, cuParams.n_thread,
                         (cuParams.n_thread / nWarp + 1) * sizeof(double) + (cuParams.n_thread / nWarp + 1) * sizeof(int) + (cuParams.n_thread / nWarp + 1) * sizeof(int)>>>(
        d_array_current_Solution,
        d_array_current_Solution_alu,
        d_array_current_Solution_col,
        d_cupoArray,
        d_alumnosSep,
        d_aluxcol,
        d_aluVulxCol,
        d_currentSolution,
        d_distMat,
        d_shuffle_students,
        d_shuffle_colegios,
        d_currentVars,
        d_choices,
        pitch);
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("3 Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("3 Async kernel error: %s\n", hipGetErrorString(errAsync));
    reduce_block_kernel<<<1, cuParams.n_block,
                          (cuParams.n_block / nWarp + 1) * sizeof(double) + (cuParams.n_block / nWarp + 1) * sizeof(int) + (cuParams.n_block / nWarp + 1) * sizeof(int)>>>(d_array_current_Solution,
                                                                                                                                                                           d_array_current_Solution_alu,
                                                                                                                                                                           d_array_current_Solution_col);
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("4 Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("4 Async kernel error: %s\n", hipGetErrorString(errAsync));
}

void CUDAWrapper::newSolutionRandomSelection(uniform_int_distribution<int> dist, uniform_int_distribution<int> dist2) {
    /********************************
    /* Metodo Nuevo
    */
    cuParams.selectThread = dist(mt);
    cuParams.selectBlock = dist2(mt);
    hipMemcpy(&d_array_current_Solution_alu[0], &cuParams.selectThread, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_array_current_Solution_col, &cuParams.selectBlock, sizeof(int), hipMemcpyHostToDevice);
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("6 Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("6 Async kernel error: %s\n", hipGetErrorString(errAsync));
}

void CUDAWrapper::newSolutionUpdate(double &costCurrentSolution) {
    calculateSolution<<<1, 1>>>(d_array_current_Solution,
                                d_array_current_Solution_alu,
                                d_array_current_Solution_col,
                                d_cupoArray,
                                d_alumnosSep,
                                d_aluxcol,
                                d_aluVulxCol,
                                d_currentSolution,
                                d_distMat,
                                pitch,
                                d_choices,
                                d_currentVars,
                                d_costCurrentSolution);

    getCurrentSolutionGpuToHost(costCurrentSolution);
    synchronizeBucle();
}

void CUDAWrapper::getCurrentSolutionGpuToHost(double &costCurrentSolution) {
    hipMemcpy(&costCurrentSolution, &d_array_current_Solution[0], sizeof(double), hipMemcpyDeviceToHost);
    errAsync = hipDeviceSynchronize();
    errSync = hipGetLastError();
    if (errSync != hipSuccess)
        printf("5 Sync kernel error: %s: %s\n", hipGetErrorName(errSync), hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("5 Async kernel error: %s: %s\n", hipGetErrorName(errAsync), hipGetErrorString(errAsync));
}

void CUDAWrapper::synchronizeBucle() {
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("6 Sync kernel error: %s: %s\n", hipGetErrorName(errSync), hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("6 Async kernel error: %s: %s\n", hipGetErrorName(errAsync), hipGetErrorString(errAsync));
}

void CUDAWrapper::copySolutionToHost(int *bestSolution, int *previousSolution) {
    hipMemcpyAsync(bestSolution, d_bestSolution, saParams.n_students * sizeof(int), hipMemcpyDeviceToHost, streams[0]);
    hipMemcpyAsync(previousSolution, d_previousSolution, saParams.n_students * sizeof(int), hipMemcpyDeviceToHost, streams[1]);
    CUDAWrapper::synchronizeBucle();
}

/*
void CUDAWrapper::mallocHostInit(double* currentVars,double *previousVars,double* bestVars){

}
*/

void CUDAWrapper::mallocHost(
    int *&previousSolution,
    int *&bestSolution,
    int *&currentSolution,
    int *&cupoArray,
    int *&alumnosSep,
    double *&matrestest,
    double *&currentVars,
    double *&previousVars,
    double *&bestVars) {
    hipHostMalloc((void **)&previousSolution, sizeof(int) * saParams.n_students);
    hipHostMalloc((void **)&bestSolution, sizeof(int) * saParams.n_students);
    hipHostMalloc((void **)&currentSolution, sizeof(int) * saParams.n_students);
    hipHostMalloc((void **)&cupoArray, sizeof(int) * saParams.n_colegios);
    hipHostMalloc((void **)&alumnosSep, sizeof(int) * saParams.n_students);
    hipHostMalloc((void **)&matrestest, sizeof(double) * saParams.n_students * saParams.n_colegios);
    hipHostMalloc((void **)&saParams.shuffle_student, sizeof(int) * saParams.n_students);
    hipHostMalloc((void **)&saParams.shuffle_colegios, sizeof(int) * saParams.n_colegios);
    hipHostMalloc((void **)&currentVars, 4 * sizeof(double));
    hipHostMalloc((void **)&previousVars, 4 * sizeof(double));
    hipHostMalloc((void **)&bestVars, 4 * sizeof(double));
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
}