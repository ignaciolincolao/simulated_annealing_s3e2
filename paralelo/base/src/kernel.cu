#include "hip/hip_runtime.h"
#include <kernel.cuh>

__constant__ int d_cupoArray[85];
__constant__ double d_alpha[3];

__global__ void newSolution_kernel(
    double *d_array_current_Solution,
    int *d_array_current_Solution_thread,
    const int n_students,
    const int n_colegios,
    const int n_thread,
    const double max_dist,
    const int* __restrict__ d_alumnosSep,
    const int totalVuln,
    const int* __restrict__ d_aluxcol,
    const int* __restrict__ d_aluVulxCol,
    const int* __restrict__ d_currentSolution,
    const double* __restrict__ d_distMat,
    const int* __restrict__ d_shuffle_students,
    const int* __restrict__ d_shuffle_colegios,
    const double* __restrict__ d_currentVars,
    size_t pitch){

    /// Shared Memory
    extern __shared__ double sharedMem[];
    int* aluxcolblock = (int*)sharedMem;
    int* aluVulxColblock = (int*)&aluxcolblock[n_colegios];
    double* solutions =(double*)&aluVulxColblock[n_colegios];
    int* solutions_thread = (int*)&solutions[n_thread];
    /// Inicializa variables en 0
    int aluchange,
            colchange,
            newSchool,
            aluVulCol= 0,
            aluNoVulCol= 0,
            totalAluCol= 0,
            myID = threadIdx.x,
            currentSchool,
            salto= n_thread,
            myCurrentAluxCol = 0,
            myCurrentAluVulxCol = 0,
            myNewAluxCol = 0,
            myNewAluVulxCol = 0;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            var1,
            var2,
            var3,
            sumDist = 0.0;
    /// Inicializa arrays
    aluchange = d_shuffle_students[blockIdx.x];
    colchange = d_shuffle_colegios[threadIdx.x];
    solutions_thread[threadIdx.x] = threadIdx.x;
    currentSchool = d_currentSolution[aluchange];
    newSchool = colchange;
    sumDist= d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];

    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    sumDist-=cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + currentSchool]);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual 
    totalcostCupo-=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela nueva
    totalcostCupo-=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2)));
    ////////////////////////////////////////////////////////////////
    /////// Realiza Movimiento
    ////////////////////////////////////////////////////////////////
    //ELimina el estudiante de la escuela actual
    myCurrentAluxCol = d_aluxcol[currentSchool]-1;
    myCurrentAluVulxCol = d_aluVulxCol[currentSchool]-d_alumnosSep[aluchange];
    //Asigna al estudiante a la nueva escuela
    myNewAluxCol = d_aluxcol[newSchool]+1;
    myNewAluVulxCol = d_aluVulxCol[newSchool]+d_alumnosSep[aluchange];

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist+=cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + newSchool]);
    // seg de la escuela actual
    totalAluCol = myCurrentAluxCol;
    aluVulCol = myCurrentAluVulxCol;
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual
    totalcostCupo+=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    
    // seg de la escuela antigua
    totalAluCol = myNewAluxCol;
    aluVulCol = myNewAluVulxCol;
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela antigua
    totalcostCupo+=cu_round_n(((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2))));


    var1 = (sumDist/n_students);
    var1= (var1/max_dist);
    //cout << var1 << "\n";
    var2 = (totalSesc/2.0);
    //cout << var2 << "\n";
    var3 = (totalcostCupo /n_colegios);
    solutions[myID] =  (double)((d_alpha[0]*var1)+(d_alpha[1]*var2)+(d_alpha[2]*var3));

    __syncthreads();
    while(salto){
        if(salto-(myID+1)>myID){
            if(currentSchool!=colchange && d_shuffle_colegios[salto-(myID+1)]!=currentSchool){
                if(solutions[myID]>solutions[salto-(myID+1)]){
                    solutions[myID]=solutions[salto-(myID+1)];
                    solutions_thread[myID]=solutions_thread[salto-(myID+1)];
                }
            }
            else{
                if(currentSchool==colchange){
                    solutions[myID]=solutions[salto-(myID+1)];
                    solutions_thread[myID]=solutions_thread[salto-(myID+1)];
                }
            }
        }
        salto = (salto/2)+(salto&(2-1));
        if(salto==1){
            salto = 0;
        }
        __syncthreads();
    }
    if(myID==0)
    {
        d_array_current_Solution[blockIdx.x] = solutions[myID];
        d_array_current_Solution_thread[blockIdx.x] = solutions_thread[myID];

    }
}

__global__ void reduce_block_kernel(
    double *d_array_current_Solution,
    int *d_array_current_Solution_thread,
    int *d_array_current_Solution_block,
    const int n_block){

    extern __shared__ double sharedMem[];
    double* solutions =(double*)sharedMem;
    int* solutions_block = (int*)&solutions[n_block];
    int* solutions_thread = (int*)&solutions_block[n_block];

    int myID = threadIdx.x;
    int salto= n_block;
    solutions[myID] = d_array_current_Solution[myID];
    solutions_thread[myID] = d_array_current_Solution_thread[myID];
    solutions_block[myID]= myID;
    __syncthreads();
    while(salto){
        if(salto-(myID+1)>myID){
            if(solutions[myID]>solutions[salto-(myID+1)]){
                solutions[myID]=solutions[salto-(myID+1)];
                solutions_thread[myID]=solutions_thread[salto-(myID+1)];
                solutions_block[myID]=solutions_block[salto-(myID+1)];
            }
        }
        salto = (salto/2)+(salto&(2-1));
        if(salto==1){
            salto = 0;
        }
        __syncthreads();
    }
    if(myID==0)
    {
        d_array_current_Solution[myID] = solutions[myID];
        d_array_current_Solution_thread[myID]= solutions_thread[myID];
        d_array_current_Solution_block[myID] = solutions_block[myID];
    }
}

__global__ void calculateSolution(
    double *d_array_current_Solution,
    int *d_array_current_Solution_thread,
    int *d_array_current_Solution_block,
    const int* __restrict__ d_shuffle_students,
    const int* __restrict__ d_shuffle_colegios,
    const int n_students,
    const int n_colegios,
    const int n_thread,
    const double max_dist,
    const int* __restrict__ d_alumnosSep,
    int totalVuln,
    int* d_aluxcol,
    int* d_aluVulxCol,
    int* d_currentSolution,
    const double* __restrict__ d_distMat,
    size_t pitch,
    double *d_currentVars,
    double *d_costCurrentSolution){

    int aluchange,
    colchange,
    newSchool,
    aluVulCol= 0,
    aluNoVulCol= 0,
    totalAluCol= 0,
    currentSchool;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            var1,
            var2,
            var3,
            sumDist = 0.0;
    /// Inicializa arrays

    aluchange = d_shuffle_students[d_array_current_Solution_block[0]];
    colchange = d_shuffle_colegios[d_array_current_Solution_thread[0]];
    currentSchool = d_currentSolution[aluchange];
    newSchool = colchange;
    
    
    sumDist= d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];


    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    sumDist-=cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + currentSchool]);

    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual 

    totalcostCupo-=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    
    totalSesc-=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela nueva

    totalcostCupo-=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2)));
    
    ////////////////////////////////////////////////////////////////
    /////// Realiza Movimiento
    ////////////////////////////////////////////////////////////////
    //ELimina el estudiante de la escuela actual
    d_aluxcol[currentSchool]-=1;
    d_aluVulxCol[currentSchool]-=d_alumnosSep[aluchange];
    //Asigna al estudiante a la nueva escuela
    d_currentSolution[aluchange] = newSchool;
    d_aluxcol[newSchool]+=1;
    d_aluVulxCol[newSchool]+=d_alumnosSep[aluchange];

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist+=cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + newSchool]);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual

    totalcostCupo+=cu_round_n((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));

    // seg de la escuela antigua
    totalAluCol = d_aluxcol[newSchool];

    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=cu_round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela antigua

    totalcostCupo+=cu_round_n(((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2))));

    d_currentVars[0] = sumDist;
    d_currentVars[1] = totalSesc;
    d_currentVars[2] = totalcostCupo;

    var1 = (sumDist/n_students);
    var1= (var1/max_dist);
    //cout << var1 << "\n";
    var2 = (totalSesc/2.0);
    //cout << var2 << "\n";
    var3 = (totalcostCupo /n_colegios);
    d_costCurrentSolution[0] =  (double)((d_alpha[0]*var1)+(d_alpha[1]*var2)+(d_alpha[2]*var3));
    d_array_current_Solution[0] = d_costCurrentSolution[0];
    if(d_array_current_Solution[0] != d_costCurrentSolution[0]){
        printf("ERRORRRRRRRRRRRR no son iguales!!!!!!!!!!!!!!!!!!\n");
        printf("%lf\n",d_array_current_Solution[0]);
        printf("%lf\n",d_costCurrentSolution[0]);

    }
}


__global__ void copyMemSolution(
    int *solution,
    int *new_solution,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        solution[i] = new_solution[i];
    }
}
__global__ void copyMemCol(
    int *col,
    int *new_col,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        col[i] = new_col[i];
    }
}
__global__ void copyVars(
    double *var,
    double *new_var){

    var[threadIdx.x] = new_var[threadIdx.x];
}

__global__ void copyCost(
    double *costCurrentSolution,
    double *new_costCurrentSolution
    ){

        costCurrentSolution[0] = new_costCurrentSolution[0];

    }

__device__ double cu_round_n(double x)
{
    double digits = pow(10.0, 16);
    return trunc(x * digits) / digits;
}
