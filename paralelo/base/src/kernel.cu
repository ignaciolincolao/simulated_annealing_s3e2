#include "hip/hip_runtime.h"
#include <kernel.cuh>

__constant__ double d_alpha[3];
__constant__ int d_n_students;
__constant__ int d_n_colegios;
__constant__ double d_max_dist;
__constant__ int d_totalVuln;

__global__ void newSolution_kernel(
    DataResult *d_array_current_Solution,
    const int* __restrict__ d_cupoArray,
    const int* __restrict__ d_alumnosSep,
    const int* __restrict__ d_aluxcol,
    const int* __restrict__ d_aluVulxCol,
    const int* __restrict__ d_currentSolution,
    const double* __restrict__ d_distMat,
    const int* __restrict__ d_shuffle_students,
    const int* __restrict__ d_shuffle_colegios,
    const double* __restrict__ d_currentVars,
    size_t pitch) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int aluchange,
            newSchool,
            aluVulCol = 0,
            aluNoVulCol = 0,
            totalAluCol = 0,
            currentSchool,
            col_solution,
            alu_solution;
    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            sumDist = 0.0,
            cost_solution;
    aluchange = d_shuffle_students[tid%d_n_students]; 
    newSchool = d_shuffle_colegios[0];
    currentSchool = d_currentSolution[aluchange];
    col_solution = newSchool;
    alu_solution = aluchange;
    d_array_current_Solution[tid].col = col_solution;
    d_array_current_Solution[tid].stu = alu_solution;
    
    sumDist = d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];
    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    sumDist -= d_distMat[aluchange * pitch / sizeof(double) + currentSchool];
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc -= fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln)));
    // costcupo escuela actual 

    
    totalcostCupo -= (double)totalAluCol * fabs((double)d_cupoArray[currentSchool] - totalAluCol) / pow(((double)d_cupoArray[currentSchool] * 0.5), 2);

    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc -= fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln)));

    // costcupo escuela nueva
    totalcostCupo -= (double)totalAluCol * fabs((double)d_cupoArray[newSchool] - totalAluCol) / pow(((double)d_cupoArray[newSchool] * 0.5), 2);

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist += d_distMat[aluchange * pitch / sizeof(double) + newSchool];
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool]-1;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluVulCol -= d_alumnosSep[aluchange];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc += fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln)));
    // costcupo escuela actual
    totalcostCupo += (double)totalAluCol * fabs((double)d_cupoArray[currentSchool] - totalAluCol) / pow(((double)d_cupoArray[currentSchool] * 0.5), 2);
    
    // seg de la escuela antigua
    totalAluCol = d_aluxcol[newSchool] + 1;
    aluVulCol = d_aluVulxCol[newSchool];
    aluVulCol += d_alumnosSep[aluchange];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc += fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln)));

    // costcupo escuela antigua
    totalcostCupo += ((double)totalAluCol * fabs((double)d_cupoArray[newSchool] - totalAluCol) / pow(((double)d_cupoArray[newSchool] * 0.5), 2));

    cost_solution = d_alpha[0] * (sumDist / (d_n_students * d_max_dist));
    cost_solution += d_alpha[1] * (totalSesc * 0.5);
    cost_solution += d_alpha[2] * (totalcostCupo / d_n_colegios);

    d_array_current_Solution[tid].costSolution  = (newSchool != currentSchool) * cost_solution + (double)(0xffffffffffffffff) * (newSchool == currentSchool);
}



__global__ void reduce_kernel(DataResult *d_array_current_Solution, int N){
    #define FULL_MASK 0xFFFFFFFF
    extern __shared__ DataResult sharedMem[];
    DataResult* solutions = (DataResult*)sharedMem;
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    DataResult val;
    val.costSolution=(double)(0xffffffffffffffff);
    val.col = -1;
    val.stu = -1;

    if (idx < N) {
        val.costSolution = d_array_current_Solution[idx].costSolution;
        val.col = d_array_current_Solution[idx].col;
        val.stu = d_array_current_Solution[idx].stu;
    }
    int warpID = threadIdx.x>>5 ;
    int threadWarp = threadIdx.x & 31;
    __syncthreads();

    // Reducción a nivel de warp, cada warp encontrara al mejor y lo dejara en la memoria compartida
    for (int salto=16; salto>0; salto>>=1){ // salto>>=1 es igual a salto/2 
        double neighbour_solution = __shfl_down_sync(FULL_MASK,val.costSolution,salto);
        int col = __shfl_down_sync(FULL_MASK,val.col,salto);
        int stu = __shfl_down_sync(FULL_MASK,val.stu,salto);
        if(neighbour_solution < val.costSolution){
            val.costSolution = neighbour_solution;
            val.col = col;
            val.stu = stu;
        }
    }

    __syncthreads();

    if (threadWarp==0){
        solutions[warpID].costSolution = val.costSolution;
        solutions[warpID].col = val.col;
        solutions[warpID].stu = val.stu;
    }
    __syncthreads();
    // Reducción entre los mejores de los warps
    DataResult val2;
    val2.costSolution=(double)(0xffffffffffffffff);
    val2.col = -1;
    val2.stu = -1;
    if(warpID == 0){
        val = (threadIdx.x < blockDim.x/32)?solutions[threadWarp]:val2;
        for(int salto=16; salto>0; salto>>=1){
            double neighbour_solution = __shfl_down_sync(FULL_MASK,val.costSolution,salto);
            int a1 = __shfl_down_sync(FULL_MASK,val.col,salto);
            int a2 = __shfl_down_sync(FULL_MASK,val.stu,salto);
            if(neighbour_solution < val.costSolution){
                val.costSolution = neighbour_solution;
                val.col = a1;
                val.stu = a2;
            }
        }
        __syncthreads();
        if(threadWarp==0){
            d_array_current_Solution[blockIdx.x].costSolution = val.costSolution;
            d_array_current_Solution[blockIdx.x].col = val.col;
            d_array_current_Solution[blockIdx.x].stu = val.stu;
        }
    }
}



__global__ void calculateSolution(
    DataResult *d_array_current_Solution,
    const int* __restrict__ d_cupoArray,
    const int* __restrict__ d_alumnosSep,
    int* d_aluxcol,
    int* d_aluVulxCol,
    int* d_currentSolution,
    const double* __restrict__ d_distMat,
    size_t pitch,
    double *d_currentVars,
    double *d_costCurrentSolution,
    int id_select){

    int aluchange,
    colchange,
    newSchool,
    aluVulCol= 0,
    aluNoVulCol= 0,
    totalAluCol= 0,
    currentSchool;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            var1,
            var2,
            var3,
            sumDist = 0.0;
    /// Inicializa arrays

    aluchange = d_array_current_Solution[id_select].stu;
    colchange = d_array_current_Solution[id_select].col;
    currentSchool = d_currentSolution[aluchange];
    //printf("%d \t %.20lf | %d %d %d \n",blockIdx.x,d_array_current_Solution[0],d_array_current_Solution_alu[0],d_array_current_Solution_col[0],currentSchool);
    newSchool = colchange;

    
    sumDist= d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];

    //printf("%lf |%lf |%lf |%lf |%d |%d \n",sumDist,totalSesc,totalcostCupo,d_array_current_Solution[0],aluchange,colchange);

    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    sumDist-=d_distMat[aluchange * pitch / sizeof(double) + currentSchool];
    //printf("%lf \n",sumDist);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=fabs((aluVulCol/(double)d_totalVuln)-(aluNoVulCol/(double)(d_n_students-d_totalVuln)));
    // costcupo escuela actual 
    //printf("%lf \n",totalSesc);
    totalcostCupo-=(double)totalAluCol*fabs((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]*0.5),2);
    //printf("%lf \n",totalcostCupo);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    
    totalSesc-=fabs((aluVulCol/(double)d_totalVuln)-(aluNoVulCol/(double)(d_n_students-d_totalVuln)));

    // costcupo escuela nueva

    totalcostCupo-=(double)totalAluCol*fabs((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]*0.5),2);
    //printf("a%d \n",newSchool);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    ////////////////////////////////////////////////////////////////
    /////// Realiza Movimiento
    ////////////////////////////////////////////////////////////////
    //ELimina el estudiante de la escuela actual
    d_aluxcol[currentSchool]-=1;
    d_aluVulxCol[currentSchool]-=d_alumnosSep[aluchange];
    //Asigna al estudiante a la nueva escuela
    d_currentSolution[aluchange] = newSchool;
    d_aluxcol[newSchool]+=1;
    d_aluVulxCol[newSchool]+=d_alumnosSep[aluchange];

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist+=d_distMat[aluchange * pitch / sizeof(double) + newSchool];
    
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=fabs((aluVulCol/(double)d_totalVuln)-(aluNoVulCol/(double)(d_n_students-d_totalVuln)));
    // costcupo escuela actual

    totalcostCupo+=(double)totalAluCol*fabs((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]*0.5),2);
    //printf("%lf \n",totalcostCupo);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    // seg de la escuela antigua
    totalAluCol = d_aluxcol[newSchool];

    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=fabs((aluVulCol/(double)d_totalVuln)-(aluNoVulCol/(double)(d_n_students-d_totalVuln)));

    // costcupo escuela antigua

    totalcostCupo+=((double)totalAluCol*fabs((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]*0.5),2));
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    d_currentVars[0] = sumDist;
    d_currentVars[1] = totalSesc;
    d_currentVars[2] = totalcostCupo;
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    var1 = (sumDist/d_n_students);
    var1= (var1/d_max_dist);
    //cout << var1 << "\n";
    var2 = (totalSesc*0.5);
    //cout << var2 << "\n";
    var3 = (totalcostCupo /d_n_colegios);
    d_costCurrentSolution[0] =  (double)((d_alpha[0]*var1)+(d_alpha[1]*var2)+(d_alpha[2]*var3));
}


__global__ void copyMemSolution(
    int *solution,
    int *new_solution,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        solution[i] = new_solution[i];
    }
}
__global__ void copyMemCol(
    int *col,
    int *new_col,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        col[i] = new_col[i];
    }
}
__global__ void copyVars(
    double *var,
    double *new_var){

    var[threadIdx.x] = new_var[threadIdx.x];
}

__global__ void copyCost(
    double *costCurrentSolution,
    double *new_costCurrentSolution
    ){

        costCurrentSolution[0] = new_costCurrentSolution[0];

    }
