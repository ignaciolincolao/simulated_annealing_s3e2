#include "hip/hip_runtime.h"
#include <kernel.cuh>

__constant__ int d_cupoArray[85];
__constant__ double d_alpha[3];

__global__ void newSolution_kernel(
    double *d_array_current_Solution,
    int *d_array_current_Solution_thread,
    const int n_students,
    const int n_colegios,
    const int n_thread,
    const double max_dist,
    const int* __restrict__ d_alumnosSep,
    const int totalVuln,
    const int* __restrict__ d_aluxcol,
    const int* __restrict__ d_aluVulxCol,
    const int* __restrict__ d_currentSolution,
    const double* __restrict__ d_distMat,
    const int* __restrict__ d_shuffle_students,
    const int* __restrict__ d_shuffle_colegios,
    size_t pitch){

/// Shared Memory
extern __shared__ double sharedMem[];
int* aluxcolblock = (int*)sharedMem;
int* aluVulxColblock = (int*)&aluxcolblock[n_colegios];
double* solutions =(double*)&aluVulxColblock[n_colegios];
int* solutions_thread = (int*)&solutions[n_thread];
/// Inicializa variables en 0
int aluchange,
        colchange,
        i = 0,
        x = 0,
        aluVulCol= 0,
        aluNoVulCol= 0,
        totalAluCol= 0,
        myID = threadIdx.x,
        school_alu_change,
        salto= n_thread;

double  totalcostCupo= 0.0,
        totalSesc= 0.0,
        var1,
        var2,
        var3,
        result= 0.0;
/// Inicializa arrays
aluchange = d_shuffle_students[blockIdx.x];
colchange = d_shuffle_colegios[threadIdx.x];
solutions_thread[threadIdx.x] = threadIdx.x;

/// Recopila la informacion que existe en memoria global
/// a shared memory
school_alu_change = d_currentSolution[aluchange];
for (i = threadIdx.x; i< n_colegios; i=i+n_thread){
    aluxcolblock[i] = d_aluxcol[i];
    aluVulxColblock[i] = d_aluVulxCol[i];
    if(i == school_alu_change){
        aluxcolblock[school_alu_change]-=1;
        aluVulxColblock[school_alu_change]-=d_alumnosSep[aluchange];
    }
}

/// Calcula la distancia total
for (x = 0 ; x < n_students ; x++) {
    if (x != aluchange) {
        result += d_distMat[x * pitch / sizeof(double) + d_currentSolution[x]];
    }
    else {
        result += d_distMat[x * pitch / sizeof(double) + colchange];
    }
}
__syncthreads();
/// Calcula el costo cupo y la cantidad de segregación total
for(int n=0; n<n_colegios; n++){
    totalAluCol = aluxcolblock[n];
    aluVulCol = aluVulxColblock[n];
    if(n == colchange){
        totalAluCol+=1;
        aluVulCol+=d_alumnosSep[aluchange];
    }
    aluNoVulCol =totalAluCol - aluVulCol;
    // Calcula el costo cupo
    totalcostCupo+=totalAluCol*fabs((d_cupoArray[n]-totalAluCol)/pow(((double)d_cupoArray[n]/2),2));
    // Calcula el total sesc
    totalSesc+=((double)1/2)*fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln)));
}

var1 = d_alpha[0]*((result/(double)n_students)/(double)max_dist);
var2 = d_alpha[1]*totalSesc;
var3 = d_alpha[2]*(totalcostCupo/n_colegios);
solutions[myID] = var1+var2+var3;

__syncthreads();
while(salto){
    if(salto-(myID+1)>myID){
        if(school_alu_change!=colchange && d_shuffle_colegios[salto-(myID+1)]!=school_alu_change){
            if(solutions[myID]>solutions[salto-(myID+1)]){
                solutions[myID]=solutions[salto-(myID+1)];
                solutions_thread[myID]=solutions_thread[salto-(myID+1)];
            }
        }
        else{
            if(school_alu_change==colchange){
                solutions[myID]=solutions[salto-(myID+1)];
                solutions_thread[myID]=solutions_thread[salto-(myID+1)];
            }
        }
    }
    salto = (salto/2)+(salto&(2-1));
    if(salto==1){
        salto = 0;
    }
    __syncthreads();
}
if(myID==0)
{
    d_array_current_Solution[blockIdx.x] = solutions[myID];
    d_array_current_Solution_thread[blockIdx.x] = solutions_thread[myID];

}
}

__global__ void reduce_block_kernel(
    double *d_array_current_Solution,
    int *d_array_current_Solution_thread,
    int *d_array_current_Solution_block,
    const int n_block){

extern __shared__ double sharedMem[];
double* solutions =(double*)sharedMem;
int* solutions_block = (int*)&solutions[n_block];
int* solutions_thread = (int*)&solutions_block[n_block];

int myID = threadIdx.x;
int salto= n_block;
solutions[myID] = d_array_current_Solution[myID];
solutions_thread[myID] = d_array_current_Solution_thread[myID];
solutions_block[myID]= myID;
__syncthreads();
while(salto){
    if(salto-(myID+1)>myID){
        if(solutions[myID]>solutions[salto-(myID+1)]){
            solutions[myID]=solutions[salto-(myID+1)];
            solutions_thread[myID]=solutions_thread[salto-(myID+1)];
            solutions_block[myID]=solutions_block[salto-(myID+1)];
        }
    }
    salto = (salto/2)+(salto&(2-1));
    if(salto==1){
        salto = 0;
    }
    __syncthreads();
}
if(myID==0)
{
    d_array_current_Solution[myID] = solutions[myID];
    d_array_current_Solution_thread[myID]= solutions_thread[myID];
    d_array_current_Solution_block[myID] = solutions_block[myID];
}
}

