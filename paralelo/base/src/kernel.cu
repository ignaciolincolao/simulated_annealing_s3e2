#include "hip/hip_runtime.h"
#include <kernel.cuh>

__constant__ double d_alpha[3];
__constant__ int d_n_students;
__constant__ int d_n_colegios;
__constant__ double d_max_dist;
__constant__ int d_totalVuln;



__global__ void newSolution_kernel(
    DataResult *d_array_current_Solution,
    const int* __restrict__ d_cupoArray,
    const int* __restrict__ d_alumnosSep,
    const int* __restrict__ d_aluxcol,
    const int* __restrict__ d_aluVulxCol,
    const int* __restrict__ d_currentSolution,
    const double* __restrict__ d_distMat,
    const int* __restrict__ d_shuffle_students,
    const int* __restrict__ d_shuffle_colegios,
    const double* __restrict__ d_currentVars,
    size_t pitch) {

    /// Inicializa variables en 0
    int aluchange,
            newSchool,
            aluVulCol = 0,
            aluNoVulCol = 0,
            totalAluCol = 0,
            currentSchool;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            sumDist = 0.0;
            
    /// Inicializa arrays
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    aluchange = d_shuffle_students[tid%d_n_students]; 
    //aluchange = d_shuffle_students[threadIdx.x]; 
    newSchool = d_shuffle_colegios[blockIdx.x%d_n_colegios];
    currentSchool = d_currentSolution[aluchange];
    //printf("%d|%d|%d|%d\n",newSchool,currentSchool,aluchange,tid%d_n_students);

    double cost_solution;
    int col_solution = newSchool;
    int alu_solution = aluchange;
    d_array_current_Solution[tid].col = col_solution;
    d_array_current_Solution[tid].stu = alu_solution;
    
    sumDist = d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];
    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia

    sumDist -= d_distMat[aluchange * pitch / sizeof(double) + currentSchool];
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc -= fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln)));
    // costcupo escuela actual 

    
    totalcostCupo -= (double)totalAluCol * fabs((double)d_cupoArray[currentSchool] - totalAluCol) / pow(((double)d_cupoArray[currentSchool] * 0.5), 2);

    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc -= fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln)));

    // costcupo escuela nueva
    totalcostCupo -= (double)totalAluCol * fabs((double)d_cupoArray[newSchool] - totalAluCol) / pow(((double)d_cupoArray[newSchool] * 0.5), 2);

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist += d_distMat[aluchange * pitch / sizeof(double) + newSchool];
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool]-1;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluVulCol -= d_alumnosSep[aluchange];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc += fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln)));
    // costcupo escuela actual
    totalcostCupo += (double)totalAluCol * fabs((double)d_cupoArray[currentSchool] - totalAluCol) / pow(((double)d_cupoArray[currentSchool] * 0.5), 2);
    
    // seg de la escuela antigua
    totalAluCol = d_aluxcol[newSchool] + 1;
    aluVulCol = d_aluVulxCol[newSchool];
    aluVulCol += d_alumnosSep[aluchange];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc += fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln)));

    // costcupo escuela antigua
    totalcostCupo += ((double)totalAluCol * fabs((double)d_cupoArray[newSchool] - totalAluCol) / pow(((double)d_cupoArray[newSchool] * 0.5), 2));

    cost_solution = d_alpha[0] * (sumDist / (d_n_students * d_max_dist));
    cost_solution += d_alpha[1] * (totalSesc * 0.5);
    cost_solution += d_alpha[2] * (totalcostCupo / d_n_colegios);

    d_array_current_Solution[tid].costSolution  = (newSchool != currentSchool) * cost_solution + (double)(0xffffffffffffffff) * (newSchool == currentSchool);
}

__global__ void calculateSolution(
    DataResult *d_array_current_Solution,
    const int* __restrict__ d_cupoArray,
    const int* __restrict__ d_alumnosSep,
    int* d_aluxcol,
    int* d_aluVulxCol,
    int* d_currentSolution,
    const double* __restrict__ d_distMat,
    size_t pitch,
    double *d_currentVars,
    double *d_costCurrentSolution,
    int id_select){

    int aluchange,
    colchange,
    newSchool,
    aluVulCol= 0,
    aluNoVulCol= 0,
    totalAluCol= 0,
    currentSchool;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            var1,
            var2,
            var3,
            sumDist = 0.0;
    /// Inicializa arrays

    aluchange = d_array_current_Solution[id_select].stu;
    colchange = d_array_current_Solution[id_select].col;
    currentSchool = d_currentSolution[aluchange];
    //printf("%d \t %.20lf | %d %d %d \n",blockIdx.x,d_array_current_Solution[0],d_array_current_Solution_alu[0],d_array_current_Solution_col[0],currentSchool);
    newSchool = colchange;

    
    sumDist= d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];

    //printf("%lf |%lf |%lf |%lf |%d |%d \n",sumDist,totalSesc,totalcostCupo,d_array_current_Solution[0],aluchange,colchange);

    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    sumDist-=d_distMat[aluchange * pitch / sizeof(double) + currentSchool];
    //printf("%lf \n",sumDist);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=fabs((aluVulCol/(double)d_totalVuln)-(aluNoVulCol/(double)(d_n_students-d_totalVuln)));
    // costcupo escuela actual 
    //printf("%lf \n",totalSesc);
    totalcostCupo-=(double)totalAluCol*fabs((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]*0.5),2);
    //printf("%lf \n",totalcostCupo);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    
    totalSesc-=fabs((aluVulCol/(double)d_totalVuln)-(aluNoVulCol/(double)(d_n_students-d_totalVuln)));

    // costcupo escuela nueva

    totalcostCupo-=(double)totalAluCol*fabs((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]*0.5),2);
    //printf("a%d \n",newSchool);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    ////////////////////////////////////////////////////////////////
    /////// Realiza Movimiento
    ////////////////////////////////////////////////////////////////
    //ELimina el estudiante de la escuela actual
    d_aluxcol[currentSchool]-=1;
    d_aluVulxCol[currentSchool]-=d_alumnosSep[aluchange];
    //Asigna al estudiante a la nueva escuela
    d_currentSolution[aluchange] = newSchool;
    d_aluxcol[newSchool]+=1;
    d_aluVulxCol[newSchool]+=d_alumnosSep[aluchange];

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist+=d_distMat[aluchange * pitch / sizeof(double) + newSchool];
    
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=fabs((aluVulCol/(double)d_totalVuln)-(aluNoVulCol/(double)(d_n_students-d_totalVuln)));
    // costcupo escuela actual

    totalcostCupo+=(double)totalAluCol*fabs((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]*0.5),2);
    //printf("%lf \n",totalcostCupo);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    // seg de la escuela antigua
    totalAluCol = d_aluxcol[newSchool];

    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=fabs((aluVulCol/(double)d_totalVuln)-(aluNoVulCol/(double)(d_n_students-d_totalVuln)));

    // costcupo escuela antigua

    totalcostCupo+=((double)totalAluCol*fabs((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]*0.5),2));
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    d_currentVars[0] = sumDist;
    d_currentVars[1] = totalSesc;
    d_currentVars[2] = totalcostCupo;
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    var1 = (sumDist/d_n_students);
    var1= (var1/d_max_dist);
    //cout << var1 << "\n";
    var2 = (totalSesc*0.5);
    //cout << var2 << "\n";
    var3 = (totalcostCupo /d_n_colegios);
    d_costCurrentSolution[0] =  (double)((d_alpha[0]*var1)+(d_alpha[1]*var2)+(d_alpha[2]*var3));
}


__global__ void copyMemSolution(
    int *solution,
    int *new_solution,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        solution[i] = new_solution[i];
    }
}
__global__ void copyMemCol(
    int *col,
    int *new_col,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        col[i] = new_col[i];
    }
}
__global__ void copyVars(
    double *var,
    double *new_var){

    var[threadIdx.x] = new_var[threadIdx.x];
}

__global__ void copyCost(
    double *costCurrentSolution,
    double *new_costCurrentSolution
    ){

        costCurrentSolution[0] = new_costCurrentSolution[0];

    }
