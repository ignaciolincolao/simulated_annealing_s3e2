#include "hip/hip_runtime.h"
#include <kernel.cuh>

__constant__ double d_alpha[4];
__constant__ int d_n_students;
__constant__ int d_n_colegios;
__constant__ double d_max_dist;
__constant__ int d_totalVuln;

__global__ void newSolution_kernel(
    double *d_array_current_Solution,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col,
    const int *__restrict__ d_cupoArray,
    const int *__restrict__ d_alumnosSep,
    const int *__restrict__ d_aluxcol,
    const int *__restrict__ d_aluVulxCol,
    const int *__restrict__ d_currentSolution,
    const double *__restrict__ d_distMat,
    const int *__restrict__ d_shuffle_students,
    const int *__restrict__ d_shuffle_colegios,
    const double *__restrict__ d_currentVars,
    size_t pitch,
    size_t penalty) {

    /// Shared Memory
    extern __shared__ double sharedMem[];
    double *solutions = (double *)sharedMem;
    int *solutions_col = (int *)&solutions[(blockDim.x >> 5) + 1]; // blockDim.x>>5  --> blockDim.x/32
    int *solutions_alu = (int *)&solutions_col[(blockDim.x >> 5) + 1];
    /// Inicializa variables en 0
    int aluchange,
        newSchool,
        aluVulCol = 0,
        aluNoVulCol = 0,
        totalAluCol = 0,
        myID = threadIdx.x,
        currentSchool;

    double totalcostCupo = 0.0,
           totalSesc = 0.0,
           sumDist = 0.0;

    /// Inicializa arrays
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    aluchange = d_shuffle_students[tid % d_n_students];
    // aluchange = d_shuffle_students[threadIdx.x];
    newSchool = d_shuffle_colegios[blockIdx.x % d_n_colegios];
    currentSchool = d_currentSolution[aluchange];
    // printf("%d|%d|%d|%d\n",newSchool,currentSchool,aluchange,tid%d_n_students);

    double cost_solution;
    int col_solution = newSchool;
    int alu_solution = aluchange;

    sumDist = d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];
    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia

    sumDist -= cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + currentSchool]);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    // cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc -= cu_round_n(fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln))));
    // costcupo escuela actual

    totalcostCupo -= cu_round_n((double)totalAluCol * fabs((double)d_cupoArray[currentSchool] - totalAluCol) / pow(((double)d_cupoArray[currentSchool] * 0.5), 2));

    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    // cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc -= cu_round_n(fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln))));

    // costcupo escuela nueva
    totalcostCupo -= cu_round_n((double)totalAluCol * fabs((double)d_cupoArray[newSchool] - totalAluCol) / pow(((double)d_cupoArray[newSchool] * 0.5), 2));

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist += cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + newSchool]);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool] - 1;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluVulCol -= d_alumnosSep[aluchange];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc += cu_round_n(fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln))));
    // costcupo escuela actual
    totalcostCupo += cu_round_n((double)totalAluCol * fabs((double)d_cupoArray[currentSchool] - totalAluCol) / pow(((double)d_cupoArray[currentSchool] * 0.5), 2));

    // seg de la escuela antigua
    totalAluCol = d_aluxcol[newSchool] + 1;
    aluVulCol = d_aluVulxCol[newSchool];
    aluVulCol += d_alumnosSep[aluchange];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc += cu_round_n(fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln))));

    // costcupo escuela antigua
    totalcostCupo += cu_round_n(((double)totalAluCol * fabs((double)d_cupoArray[newSchool] - totalAluCol) / pow(((double)d_cupoArray[newSchool] * 0.5), 2)));

    cost_solution = d_alpha[0] * (sumDist / (d_n_students * d_max_dist));
    cost_solution += d_alpha[1] * (totalSesc * 0.5);
    cost_solution += d_alpha[2] * (totalcostCupo / d_n_colegios);
    cost_solution += d_alpha[3] * penalty;
    // printf("%.16lf %d %d\n",solutions[myID], colchange,aluchange);
    __syncthreads();

    // Encuentra el minimo
    int warpID = threadIdx.x / 32;
    int lane = threadIdx.x % 32;
#define FULL_MASK 0xffffffff
    // Encuentra minimo a nivel de warp
    for (int salto = 32 / 2; salto > 0; salto >>= 1) { // salto>>=1 es igual a salto/2
        /*
        double neightbour_solutions[2] = {cost_solution, __shfl_down_sync(FULL_MASK,cost_solution,salto)};
        int cols_solutions[2] = {col_solution, __shfl_down_sync(FULL_MASK,col_solution,salto)};
        int alus_solutions[2] = {alu_solution, __shfl_down_sync(FULL_MASK,alu_solution,salto)};
        int pos = (neightbour_solutions[1] < neightbour_solutions[0]);
        cost_solution = neightbour_solutions[pos];
        col_solution = cols_solutions[pos];
        alu_solution = alus_solutions[pos];
        */

        double neighbour_solution = __shfl_down_sync(FULL_MASK, cost_solution, salto);
        int co = __shfl_down_sync(FULL_MASK, col_solution, salto);
        int al = __shfl_down_sync(FULL_MASK, alu_solution, salto);
        if (neighbour_solution < cost_solution) {
            cost_solution = neighbour_solution;
            col_solution = co;
            alu_solution = al;
        }
    }
    if (lane == 0) {
        solutions[warpID] = cost_solution;
        solutions_col[warpID] = col_solution;
        solutions_alu[warpID] = alu_solution;
    }

    __syncthreads();
    // Encuentra el minimo a nivel de bloque
    if (warpID == 0) {
        /*
        int select= myID < blockDim.x/32;
        double cost_solution_select[2] = {9999.9,solutions[lane]};

        cost_solution = cost_solution_select[select];
        col_solution = solutions_col[lane];
        alu_solution = solutions_alu[lane];
        */

        cost_solution = (myID < blockDim.x / 32) ? solutions[lane] : 9999;
        col_solution = (myID < blockDim.x / 32) ? solutions_col[lane] : -1;
        alu_solution = (myID < blockDim.x / 32) ? solutions_alu[lane] : -1;
        // printf("laneID= %d blockIdx.x= %d | %.16lf %d %d\n",lane, blockIdx.x, cost_solution, alu_solution, col_solution);
        for (int salto = 32 / 2; salto > 0; salto >>= 1) {
            /*
            double neightbour_solutions[2] = {cost_solution, __shfl_down_sync(FULL_MASK,cost_solution,salto)};
            int cols_solutions[2] = {col_solution, __shfl_down_sync(FULL_MASK,col_solution,salto)};
            int alus_solutions[2] = {alu_solution, __shfl_down_sync(FULL_MASK,alu_solution,salto)};
            int pos = (neightbour_solutions[1] < neightbour_solutions[0]);
            cost_solution = neightbour_solutions[pos];
            col_solution = cols_solutions[pos];
            alu_solution = alus_solutions[pos];
            */
            double neighbour_solution = __shfl_down_sync(FULL_MASK, cost_solution, salto);
            int co = __shfl_down_sync(FULL_MASK, col_solution, salto);
            int al = __shfl_down_sync(FULL_MASK, alu_solution, salto);
            if (neighbour_solution < cost_solution) {
                cost_solution = neighbour_solution;
                col_solution = co;
                alu_solution = al;
            }
        }
        __syncthreads();
        if (lane == 0) {
            d_array_current_Solution[blockIdx.x] = cost_solution;
            d_array_current_Solution_alu[blockIdx.x] = alu_solution;
            d_array_current_Solution_col[blockIdx.x] = col_solution;
            // printf("laneID= %d %.16lf %d %d\n",lane, cost_solution, alu_solution, col_solution);
        }
    }
}

__global__ void reduce_block_kernel(
    double *d_array_current_Solution,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col) {

    extern __shared__ double sharedMem[];
    double *solutions = (double *)sharedMem;
    int *solutions_col = (int *)&solutions[blockDim.x / 32 + 1];
    int *solutions_alu = (int *)&solutions_col[blockDim.x / 32 + 1];
    int myID = threadIdx.x;
    int end = blockDim.x - 1;

    double cost_solution = d_array_current_Solution[myID];
    int col_solution = d_array_current_Solution_col[myID];
    int alu_solution = d_array_current_Solution_alu[myID];
    int warpID = threadIdx.x / 32;
    int lane = threadIdx.x % 32;
#define FULL_MASK 0xffffffff

    if (myID == 0) {
        if (d_array_current_Solution[end] < cost_solution) {
            cost_solution = d_array_current_Solution[end];
            col_solution = d_array_current_Solution_col[end];
            alu_solution = d_array_current_Solution_alu[end];
        }
    }

    // Encuentra minimo a nivel de warp
    // printf("%.16lf %d %d\n", cost_solution,col_solution,alu_solution);
    for (int salto = 32 / 2; salto > 0; salto >>= 1) { // salto>>=1 es igual a salto/2
        /*
        double neightbour_solutions[2] = {cost_solution, __shfl_down_sync(FULL_MASK,cost_solution,salto)};
        int cols_solutions[2] = {col_solution, __shfl_down_sync(FULL_MASK,col_solution,salto)};
        int alus_solutions[2] = {alu_solution, __shfl_down_sync(FULL_MASK,alu_solution,salto)};
        int pos = (neightbour_solutions[1] < neightbour_solutions[0]);
        cost_solution = neightbour_solutions[pos];
        col_solution = cols_solutions[pos];
        alu_solution = alus_solutions[pos];
        */

        double neighbour_solution = __shfl_down_sync(FULL_MASK, cost_solution, salto);
        int co = __shfl_down_sync(FULL_MASK, col_solution, salto);
        int al = __shfl_down_sync(FULL_MASK, alu_solution, salto);
        if (neighbour_solution < cost_solution) {
            cost_solution = neighbour_solution;
            col_solution = co;
            alu_solution = al;
        }
    }
    if (lane == 0) {
        solutions[warpID] = cost_solution;
        solutions_col[warpID] = col_solution;
        solutions_alu[warpID] = alu_solution;
    }

    __syncthreads();
    // Encuentra el minimo a nivel de bloque
    if (warpID == 0) {
        /*
        int select= myID < blockDim.x/32;
        double cost_solution_select[2] = {9999.9,solutions[lane]};

        cost_solution = cost_solution_select[select];
        col_solution = solutions_col[lane];
        alu_solution = solutions_alu[lane];
        */
        cost_solution = (myID < blockDim.x / 32) ? solutions[lane] : 9999;
        col_solution = (myID < blockDim.x / 32) ? solutions_col[lane] : -1;
        alu_solution = (myID < blockDim.x / 32) ? solutions_alu[lane] : -1;
        // printf("laneID= %d %.16lf %d %d\n",lane, cost_solution, alu_solution, col_solution);
        for (int salto = 32 / 2; salto > 0; salto >>= 1) {
            /*
            double neightbour_solutions[2] = {cost_solution, __shfl_down_sync(FULL_MASK,cost_solution,salto)};
            int cols_solutions[2] = {col_solution, __shfl_down_sync(FULL_MASK,col_solution,salto)};
            int alus_solutions[2] = {alu_solution, __shfl_down_sync(FULL_MASK,alu_solution,salto)};
            int pos = (neightbour_solutions[1] < neightbour_solutions[0]);
            cost_solution = neightbour_solutions[pos];
            col_solution = cols_solutions[pos];
            alu_solution = alus_solutions[pos];
            */
            double neighbour_solution = __shfl_down_sync(FULL_MASK, cost_solution, salto);
            int co = __shfl_down_sync(FULL_MASK, col_solution, salto);
            int al = __shfl_down_sync(FULL_MASK, alu_solution, salto);
            if (neighbour_solution < cost_solution) {
                cost_solution = neighbour_solution;
                col_solution = co;
                alu_solution = al;
            }
        }
        //__syncthreads();
        if (lane == 0) {
            d_array_current_Solution[blockIdx.x] = cost_solution;
            d_array_current_Solution_alu[blockIdx.x] = alu_solution;
            d_array_current_Solution_col[blockIdx.x] = col_solution;
            // printf("laneID= %d %.16lf %d %d\n",lane, cost_solution, alu_solution, col_solution);
        }
    }
}

__global__ void reduce_block_max(
    double *d_array_current_Solution,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col) {

    extern __shared__ double sharedMem[];
    double *solutions = (double *)sharedMem;
    int *solutions_col = (int *)&solutions[blockDim.x / 32 + 1];
    int *solutions_alu = (int *)&solutions_col[blockDim.x / 32 + 1];
    int myID = threadIdx.x;
    int end = blockDim.x - 1;

    double cost_solution = d_array_current_Solution[myID];
    int col_solution = d_array_current_Solution_col[myID];
    int alu_solution = d_array_current_Solution_alu[myID];
    int warpID = threadIdx.x / 32;
    int lane = threadIdx.x % 32;
#define FULL_MASK 0xffffffff

    if (myID == 0) {
        if (d_array_current_Solution[end] < cost_solution) {
            cost_solution = d_array_current_Solution[end];
            col_solution = d_array_current_Solution_col[end];
            alu_solution = d_array_current_Solution_alu[end];
        }
    }

    // Encuentra minimo a nivel de warp
    // printf("%.16lf %d %d\n", cost_solution,col_solution,alu_solution);
    for (int salto = 32 / 2; salto > 0; salto >>= 1) { // salto>>=1 es igual a salto/2
        double neightbour_solutions[2] = {cost_solution, __shfl_down_sync(FULL_MASK, cost_solution, salto)};
        int cols_solutions[2] = {col_solution, __shfl_down_sync(FULL_MASK, col_solution, salto)};
        int alus_solutions[2] = {alu_solution, __shfl_down_sync(FULL_MASK, alu_solution, salto)};
        int pos = (neightbour_solutions[1] < neightbour_solutions[0]);
        cost_solution = neightbour_solutions[pos];
        col_solution = cols_solutions[pos];
        alu_solution = alus_solutions[pos];
        /*
        double neighbour_solution = __shfl_down_sync(FULL_MASK,cost_solution,salto);
        int co = __shfl_down_sync(FULL_MASK,col_solution,salto);
        int al = __shfl_down_sync(FULL_MASK,alu_solution,salto);
        if(neighbour_solution < cost_solution){
            cost_solution = neighbour_solution;
            col_solution = co;
            alu_solution = al;
        }
        */
    }
    if (lane == 0) {
        solutions[warpID] = cost_solution;
        solutions_col[warpID] = col_solution;
        solutions_alu[warpID] = alu_solution;
    }

    __syncthreads();
    // Encuentra el minimo a nivel de bloque
    if (warpID == 0) {
        cost_solution = (myID < blockDim.x / 32) ? solutions[lane] : 0.000000000000;
        col_solution = (myID < blockDim.x / 32) ? solutions_col[lane] : 0;
        alu_solution = (myID < blockDim.x / 32) ? solutions_alu[lane] : 0;
        // printf("laneID= %d %.16lf %d %d\n",lane, cost_solution, alu_solution, col_solution);
        for (int salto = 32 / 2; salto > 0; salto >>= 1) {
            double neightbour_solutions[2] = {cost_solution, __shfl_down_sync(FULL_MASK, cost_solution, salto)};
            int cols_solutions[2] = {col_solution, __shfl_down_sync(FULL_MASK, col_solution, salto)};
            int alus_solutions[2] = {alu_solution, __shfl_down_sync(FULL_MASK, alu_solution, salto)};
            int pos = (neightbour_solutions[1] < neightbour_solutions[0]);
            cost_solution = neightbour_solutions[pos];
            col_solution = cols_solutions[pos];
            alu_solution = alus_solutions[pos];
            /*
            double neighbour_solution = __shfl_down_sync(FULL_MASK,cost_solution,salto);
            int co = __shfl_down_sync(FULL_MASK,col_solution,salto);
            int al = __shfl_down_sync(FULL_MASK,alu_solution,salto);
            if(neighbour_solution < cost_solution){
                cost_solution = neighbour_solution;
                col_solution = co;
                alu_solution = al;
            }
            */
        }
        //__syncthreads();
        if (lane == 0) {
            d_array_current_Solution[blockIdx.x] = cost_solution;
            d_array_current_Solution_alu[blockIdx.x] = alu_solution;
            d_array_current_Solution_col[blockIdx.x] = col_solution;
            // printf("laneID= %d %.16lf %d %d\n",lane, cost_solution, alu_solution, col_solution);
        }
    }
}

__global__ void calculateSolution(
    double *d_array_current_Solution,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col,
    const int *__restrict__ d_cupoArray,
    const int *__restrict__ d_alumnosSep,
    int *d_aluxcol,
    int *d_aluVulxCol,
    int *d_currentSolution,
    const double *__restrict__ d_distMat,
    size_t pitch,
    size_t penalty,
    double *d_currentVars,
    double *d_costCurrentSolution) {

    int aluchange,
        colchange,
        newSchool,
        aluVulCol = 0,
        aluNoVulCol = 0,
        totalAluCol = 0,
        currentSchool;

    double totalcostCupo = 0.0,
           totalSesc = 0.0,
           var1,
           var2,
           var3,
           sumDist = 0.0;

    size_t var4;
    /// Inicializa arrays

    aluchange = d_array_current_Solution_alu[0];
    colchange = d_array_current_Solution_col[0];
    currentSchool = d_currentSolution[aluchange];
    // printf("%d \t %.20lf | %d %d %d \n",blockIdx.x,d_array_current_Solution[0],d_array_current_Solution_alu[0],d_array_current_Solution_col[0],currentSchool);
    newSchool = colchange;

    sumDist = d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];
    // printf("%lf |%lf |%lf |%lf |%d |%d \n",sumDist,totalSesc,totalcostCupo,d_array_current_Solution[0],aluchange,colchange);

    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    sumDist -= cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + currentSchool]);
    // printf("%lf \n",sumDist);
    //  seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];

    // cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc -= cu_round_n(fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln))));
    // costcupo escuela actual
    // printf("%lf \n",totalSesc);
    totalcostCupo -= cu_round_n((double)totalAluCol * fabs((double)d_cupoArray[currentSchool] - totalAluCol) / pow(((double)d_cupoArray[currentSchool] * 0.5), 2));
    // printf("%lf \n",totalcostCupo);
    // printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    //  seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    // cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol = totalAluCol - aluVulCol;

    totalSesc -= cu_round_n(fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln))));

    // costcupo escuela nueva

    totalcostCupo -= cu_round_n((double)totalAluCol * fabs((double)d_cupoArray[newSchool] - totalAluCol) / pow(((double)d_cupoArray[newSchool] * 0.5), 2));
    // printf("a%d \n",newSchool);
    // printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    ////////////////////////////////////////////////////////////////
    /////// Realiza Movimiento
    ////////////////////////////////////////////////////////////////
    // ELimina el estudiante de la escuela actual
    d_aluxcol[currentSchool] -= 1;
    d_aluVulxCol[currentSchool] -= d_alumnosSep[aluchange];
    // Asigna al estudiante a la nueva escuela
    d_currentSolution[aluchange] = newSchool;
    d_aluxcol[newSchool] += 1;
    d_aluVulxCol[newSchool] += d_alumnosSep[aluchange];

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist += cu_round_n(d_distMat[aluchange * pitch / sizeof(double) + newSchool]);

    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc += cu_round_n(fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln))));
    // costcupo escuela actual

    totalcostCupo += cu_round_n((double)totalAluCol * fabs((double)d_cupoArray[currentSchool] - totalAluCol) / pow(((double)d_cupoArray[currentSchool] * 0.5), 2));
    // printf("%lf \n",totalcostCupo);
    // printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    //  seg de la escuela antigua
    totalAluCol = d_aluxcol[newSchool];

    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol = totalAluCol - aluVulCol;
    totalSesc += cu_round_n(fabs((aluVulCol / (double)d_totalVuln) - (aluNoVulCol / (double)(d_n_students - d_totalVuln))));

    // costcupo escuela antigua

    totalcostCupo += cu_round_n(((double)totalAluCol * fabs((double)d_cupoArray[newSchool] - totalAluCol) / pow(((double)d_cupoArray[newSchool] * 0.5), 2)));
    // printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    d_currentVars[0] = sumDist;
    d_currentVars[1] = totalSesc;
    d_currentVars[2] = totalcostCupo;
    // printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    var1 = (sumDist / d_n_students);
    var1 = (var1 / d_max_dist);
    // cout << var1 << "\n";
    var2 = (totalSesc * 0.5);
    // cout << var2 << "\n";
    var3 = (totalcostCupo / d_n_colegios);
    // std::cout << var4 << "\n";
    var4 = penalty;

    d_costCurrentSolution[0] = (double)((d_alpha[0] * var1) + (d_alpha[1] * var2) + (d_alpha[2] * var3) + (d_alpha[3] * var4));
    d_array_current_Solution[0] = d_costCurrentSolution[0];
    if (d_array_current_Solution[0] != d_costCurrentSolution[0]) {
        printf("ERRORRRRRRRRRRRR no son iguales!!!!!!!!!!!!!!!!!!\n");
        printf("%.10lf\n", d_array_current_Solution[0]);
        printf("%.10lf\n", d_costCurrentSolution[0]);
        printf("%.10lf\n", var1);
        printf("%.10lf\n", var2);
        printf("%.10lf\n", var3);
        printf("%.10lf\n", d_alpha[0]);
        printf("%.10lf\n", d_alpha[1]);
        printf("%.10lf\n", d_alpha[2]);
    }
    // d_array_current_Solution[0] = d_costCurrentSolution[0];
}

__global__ void copyMemSolution(
    int *solution,
    int *new_solution,
    int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        solution[i] = new_solution[i];
    }
}
__global__ void copyMemCol(
    int *col,
    int *new_col,
    int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        col[i] = new_col[i];
    }
}
__global__ void copyVars(
    double *var,
    double *new_var) {

    var[threadIdx.x] = new_var[threadIdx.x];
}

__global__ void copyCost(
    double *costCurrentSolution,
    double *new_costCurrentSolution) {

    costCurrentSolution[0] = new_costCurrentSolution[0];
}

inline __device__ double cu_round_n(double x) {
    // pow(10.0, 16);
    double digits = 10000000000000000;
    return trunc(x * digits) / digits;
}

__device__ size_t calcPenalty(double *currentSolution) {
}
