#include "hip/hip_runtime.h"
#include <sas-old.cuh>
#include <kernel.cuh>
#include <ExplorationCriterion.hpp>
#include <AcceptanceCriterion.hpp>
#include <TemperatureLength.hpp>
#include <ReheatingMethods.hpp>
#include <CoolingScheme.hpp>

#include <limits>

#define DECIMAL 16

#include <assert.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
///////////////////////////////////////////////////
/// Variables constantes CUDA
///////////////////////////////////////////////////



typedef std::numeric_limits<double> dbl;

///////////////////////////////////////////////////
/// Variables globales.
///////////////////////////////////////////////////




double sasFunc() {
    int x = 0, z = 0;
    int totalVuln = 0;
    cout.precision(dbl::max_digits10);
    //cout << fixed << setprecision(70) << endl;
    //srand(time(NULL));
    ///////////////////////////////////////////////////
    /// Genera archivo de almacenamiento de datos
    ///////////////////////////////////////////////////

    /*
    * Prepara archivos para guardar los datos
    */
    
    ofstream info;
    string infotxt = ruta_save + prefijo_save +"-info.txt"; 
    info.open(infotxt);
    /*
    * 
    */
    ofstream info_test;
    string nameinfo_test = ruta_save + prefijo_save+"-info-test.txt"; 
    info_test.open(nameinfo_test);
    /*
    * Genera los archivos que contienen información de los estados de estudiantes y escuelas durante
    * la ejecución del algoritmo
    */
    ofstream info_graficos;
    string name_info_graficos = ruta_save + prefijo_save +"-info-graficos.txt";
    info_graficos.open(name_info_graficos);
    ///////////////////////////////////////////////////
    /// Datos colegios
    /// Lee el archivo linea por linea y luego lo agrega al arreglo de estructura Info_colegio
    ///////////////////////////////////////////////////
    Info_colegio *ptr_colegios;
    vector<Info_colegio> colegios;
    getDataSchool(colegios);
    ptr_colegios = colegios.data();
    n_colegios = colegios.size();

    ///////////////////////////////////////////////////
    /// Datos Alumnos
    /// Lee el archivo linea por linea y luego lo agrega al arreglo de estructura info_student
    ///////////////////////////////////////////////////
    Info_alu *ptr_students;
    vector<Info_alu> students;
    getDataStudents(students,totalVuln);
    ptr_students = students.data();
    n_students = students.size();

    ///////////////////////////////////////////////////
    /// Inicializa Variables y arreglos
    ///////////////////////////////////////////////////


    int aluVulxCol[n_colegios], aluxcol[n_colegios];
    int previousAluxCol[n_colegios];
    int previousAluVulxCol[n_colegios];
    int bestAluxCol[n_colegios];
    int bestAluVulxCol[n_colegios];

    int *previousSolution = nullptr;
    int *bestSolution = nullptr;
    int *currentSolution = nullptr;
    double **distMat = nullptr;
    int *cupoArray = nullptr;
    int *alumnosSep = nullptr;

    
    double  costBestSolution,
        costPreviousSolution,
        costCurrentSolution,
        *ptr_alpha = &alpha[0];
    
    int count = 0;

    hipHostMalloc((void**)&previousSolution, sizeof(int)*n_students);
    hipHostMalloc((void**)&bestSolution, sizeof(int)*n_students);
    hipHostMalloc((void**)&currentSolution, sizeof(int)*n_students);
    hipHostMalloc((void**)&cupoArray, sizeof(int)*n_colegios);
    hipHostMalloc((void**)&alumnosSep, sizeof(int)*n_students);
    /*
    previousSolution = (int *)malloc(sizeof(int)*n_students);
    bestSolution=(int *)malloc(sizeof(int)*n_students);
    currentSolution=(int *)malloc(sizeof(int)*n_students);
    cupoArray=(int *)malloc(sizeof(int)*n_colegios);
    alumnosSep = (int *)malloc( sizeof(int)*n_students);
    */
    distMat=(double **)malloc(sizeof(double)*n_students);
    for(x=0; x < n_students; x++) {
        distMat[ x ]=(double *)malloc(sizeof(double)*n_colegios);
    }

    ///////////////////////////////////////////////////
    /// Se asignan las escuelas un arreglo que y estudiantes a la escuela
    /// las escuelas tendran como identificación el indice
    /// y currentSolution tiene como indice al estudiante y el valor del indice a la escuela que asignada
    ///////////////////////////////////////////////////
    initializeArray(aluxcol, 
                previousAluxCol, 
                bestAluxCol, 
                aluVulxCol, 
                previousAluVulxCol, 
                bestAluVulxCol, 
                alumnosSep,
                students,
                colegios);
    assignSchoolToArray(previousSolution, bestSolution, currentSolution, ptr_colegios, ptr_students, cupoArray);
    calcDist(ptr_colegios, ptr_students, distMat);
    max_dist = getMaxDistance(distMat);
    normalizedAlpha(alpha);

    ///////////////////////////////////////////////////
    /// Registro de datos
    ///////////////////////////////////////////////////
    costBestSolution = calCosto(currentSolution,distMat,ptr_alpha, alumnosSep, totalVuln, cupoArray);
    costPreviousSolution = costBestSolution;
    costCurrentSolution = costBestSolution;
    
    
    cout << "--------------- Primeros datos -------------" << "\n";
    cout << "Primer costo de solución: " << costBestSolution << "\n";
    cout << "Primer distancia: " << meanDist(currentSolution,distMat) << "\n";
    cout << "Primer Segregación: " << S(currentSolution, alumnosSep, totalVuln) << "\n";
    cout << "Primer CostoCupo: " << costCupo(currentSolution,cupoArray) << "\n";

    info << "--------------- Primeros datos -------------" << "\n";
    info << "Primer costo de solución: " << costBestSolution << "\n";
    info << "Primer distancia: " << meanDist(currentSolution,distMat) << "\n";
    info << "Primer Segregación: " << S(currentSolution, alumnosSep, totalVuln) << "\n";
    info << "Primer CostoCupo: " << costCupo(currentSolution,cupoArray) << "\n";


    ///////////////////////////////////////////////////
    /// Generación de archivos que almacenan información de los graficos
    ///////////////////////////////////////////////////

    info_graficos << setprecision(13);
    info_graficos << count << "," 
                << meanDist(currentSolution,distMat)/max_dist << "," // Distancia promedio recorrida por los estudiantes normalizada
                << meanDist(currentSolution,distMat) << "," // Distancia promedio recorrida por los estudiantes
                << S(currentSolution, alumnosSep, totalVuln) << "," // Indice de duncan
                << costCupo(currentSolution,cupoArray) << "," // Costo cupo de las escuelas
                << costCurrentSolution << "," // Solución actual
                << temp << setprecision(13) << "\n"; // Temperatura actual

    count++;
    ///////////////////////////////////////////////////
    /// Genera arreglos que contendran valores del 0 hasta n_students y n_colegios
    ///////////////////////////////////////////////////

    int *shuffle_student, *shuffle_colegios;
    hipHostMalloc((void**)&shuffle_student, sizeof(int)*n_students);
    hipHostMalloc((void**)&shuffle_colegios, sizeof(int)*n_colegios);
    for (int i = 0; i < n_students; i++) {
        shuffle_student[i] = i;
    }
    for (int i=0; i < n_colegios; i++){
        shuffle_colegios[i]=i;
    }
    ///////////////////////////////////////////////////
    /// Posicion estudiantes
    ///////////////////////////////////////////////////

    ofstream info_graficos_bestSolution;
    string name_info_graficos_bestSolution = ruta_save + prefijo_save +"-info-graficos_bestSolution.txt"; // concatenar
    info_graficos_bestSolution.open(name_info_graficos_bestSolution);
    for(x = 0; x < n_students; x++){
        info_graficos_bestSolution << currentSolution[x] << ",";
    }
    info_graficos_bestSolution << "\n";

    ///////////////////////////////////////////////////
    /// Genera distribuciones para seleccionar un estudiante y una escuela al azar
    ///////////////////////////////////////////////////

    dist = uniform_int_distribution<int>(0, n_students-1);
    dist2 = uniform_int_distribution<int>(0, n_colegios-1);

    ///////////////////////////////////////////////////
    /// Inicio el contador de tiempo antes de iniciar el algortimo
    ///////////////////////////////////////////////////
    auto start = std::chrono::high_resolution_clock::now();
    ///////////////////////////////////////////////////
    /// Comienza a ejecutarse el algoritmo de SA
    ///////////////////////////////////////////////////


    vector<double> vector_costCurrentSolution;
    vector<double> vector_meanDist;
    vector<double> vector_segregation;
    vector<double> vector_costoCupo;
    vector<double> vector_temp;
    vector<int> vector_count;


    std::vector<double> vector_historyCostSolution;
    std::vector<double> vector_historyTemp;
    std::vector<double> vector_historymeanDist;
    std::vector<double> vector_historymeanDistNorm;
    std::vector<double> vector_historySegregation;
    std::vector<double> vector_historycostoCupo;
    std::vector<bool> vector_historyAcceptSolution;
    std::vector<int> vector_historyAsign;
    std::vector<std::tuple <int,int>> vector_historyMove;


    
    int count_rechaso=0;
    // int reheating = 0;
    int c_accepta = 0;
    int c_cooling_temperature = 0;
    // int valmaxheating=n_colegios;
    // int count_reheating = 0;
    // double bestTemp = 0;
    double k_reheating_init = k_reheating;
    double temp_init = temp;
    int count_trials = 0;
    float len1_init = len1;
    float len2_init = len2;
    double len3_init = len3;
    double len4_init = len4;

    ////////////////////////////////////////////////////////////////////////
    // VARIABLES DE PRUEBA
    ////////////////////////////////////////////////////////////////////////

    // double costCurrentSolutionV2 = costCurrentSolution;
    double *currentVars;
    hipHostMalloc( (void**)&currentVars,3 * sizeof(double));
    double *previousVars;
    hipHostMalloc( (void**)&previousVars,3 * sizeof(double)); 
    double *bestVars;
    hipHostMalloc( (void**)&bestVars,3 * sizeof(double)); 

    currentVars[0] = sumDist(currentSolution,distMat);
    currentVars[1] = sumS(currentSolution, alumnosSep, totalVuln);
    currentVars[2] = sumCostCupo(currentSolution,cupoArray);
    previousVars[0] = currentVars[0];
    previousVars[1] = currentVars[1];
    previousVars[2] = currentVars[2];
    double var1,var2,var3;
    cout << costBestSolution << endl;
    var1 = (currentVars[0]/n_students);
    var1= (var1/max_dist);
    //cout << var1 << "\n";
    var2 = (currentVars[1]/2.0);
    //cout << var2 << "\n";
    var3 = (currentVars[2] /n_colegios);
    costBestSolution = (double)((ptr_alpha[0]*var1)+(ptr_alpha[1]*var2)+(ptr_alpha[2]*var3));
    cout << costBestSolution << endl;
    costPreviousSolution = costBestSolution;
    costCurrentSolution = costBestSolution;
    auto start_compare = std::chrono::high_resolution_clock::now();
    auto end_compare = std::chrono::high_resolution_clock::now();
    double time_taken_v1 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_compare - start_compare).count();
    //double time_taken_v2;
    double vector_time1 =0;
    // double vector_time2=0;

    /////////////////////////////////////////////////////////////////////////
    // VARIABLES PARA CUDA
    ////////////////////////////////////////////////////////////////////////

    double *d_distMat; /// clon de matriz de distancia
    int *d_currentSolution, *d_bestSolution, *d_previousSolution;
    int *d_alumnosSep; // Array que contendra a los estudiantes vulnerables
    int *d_cupoArray;
    ///////////////
    double *d_array_current_Solution;
    int *d_array_current_Solution_alu;
    int *d_array_current_Solution_col;
    ///////////////
    int *d_aluxcol,*d_previousAluxcol;
    int *d_aluVulxCol,*d_previousAluVulxCol;
    int *d_shuffle_students;
    int *d_shuffle_colegios;
    double *d_currentVars, *d_bestVars, *d_previousVars;
    double *d_costPreviousSolution, *d_costBestSolution, *d_costCurrentSolution;

    int max_changes_students = min(n_thread*n_block, n_students);
    int max_changes_school = min(n_block, n_colegios);



    hipMalloc((void **) &d_array_current_Solution, n_block * sizeof(double));
    hipMalloc((void **) &d_costCurrentSolution, 1 * sizeof(double));
    hipMalloc((void **) &d_costBestSolution, 1 * sizeof(double));
    hipMalloc((void **) &d_costPreviousSolution, 1 * sizeof(double));
    hipMalloc((void **) &d_currentVars, 3 * sizeof(double));
    hipMalloc((void **) &d_bestVars, 3 * sizeof(double));
    hipMalloc((void **) &d_previousVars, 3 * sizeof(double));
    hipMalloc((void **) &d_array_current_Solution_alu, n_block * sizeof(int)); 
    hipMalloc((void **) &d_array_current_Solution_col, n_block * sizeof(int));
    hipMalloc((void **) &d_shuffle_colegios, max_changes_school  * sizeof(int));
    hipMalloc((void **) &d_shuffle_students, max_changes_students * sizeof(int));
    hipMalloc((void **) &d_aluxcol,n_colegios * sizeof(int));
    hipMalloc((void **) &d_previousAluxcol,n_colegios * sizeof(int));
    hipMalloc((void **) &d_aluVulxCol,n_colegios * sizeof(int));
    hipMalloc((void **) &d_previousAluVulxCol,n_colegios * sizeof(int));
    hipMalloc((void **) &d_currentSolution, n_students * sizeof(int));  // Solución actual
    hipMalloc((void **) &d_bestSolution, n_students * sizeof(int));
    hipMalloc((void **) &d_previousSolution, n_students * sizeof(int));
    hipMalloc((void **) &d_alumnosSep, n_students * sizeof(int)); // arreglo que contiene la id de cada usuario vulnerable
    hipMalloc((void **) &d_cupoArray, n_colegios * sizeof(int));


    double *matrestest;
    hipHostMalloc( (void**)&matrestest,sizeof(double) * n_students * n_colegios); 
    double *array_costCurrentSolution = (double *) malloc(sizeof(double) * n_block * n_thread);
    for (x = 0; x < n_students; x++) {
        for (z = 0; z < n_colegios; z++) {
            matrestest[n_colegios * x + z] = distMat[x][z];
        }
    }
    for (x = 0; x < n_block; x++){
        for (z = 0; z < n_thread; z++){
            array_costCurrentSolution[n_thread * x + z] = 0.0;
        }
    }



    ////////////////////////////////////////////////////
    /////// Stream 
    ///////////////////////////////////////////////////
    int deviceId;
    int numberOfSMs;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId); // Calcula el numero de SMstream 
    hipGetDeviceProperties(&deviceProp, 0);
    int threadsPerBlock = 256;
    int numberOfBlocks = 32 * numberOfSMs;
    int NUM_STREAMS = 10;
    int nWarp = deviceProp.warpSize;
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) { hipStreamCreate(&streams[i]); }

   

    ///////////////////////////////////////////////////
    /// Valores que nunca van a cambiar
    //////////////////////////////////////////////////////




    size_t pitch;
    hipMallocPitch(&d_distMat,
                    &pitch,
                    n_colegios * sizeof(double),
                    n_students); // Reserva memoria para la matriz de distancia


    gpuErrchk( hipMemcpyToSymbolAsync(HIP_SYMBOL( d_alpha), alpha,  3 * sizeof(double),0,hipMemcpyHostToDevice,streams[2]));
    

    gpuErrchk( hipMemcpyToSymbolAsync(HIP_SYMBOL( d_n_students), &n_students, sizeof(int),0,hipMemcpyHostToDevice,streams[3]));
    gpuErrchk( hipMemcpyToSymbolAsync(HIP_SYMBOL( d_n_colegios), &n_colegios, sizeof(int),0,hipMemcpyHostToDevice,streams[4]));
    gpuErrchk( hipMemcpyToSymbolAsync(HIP_SYMBOL( d_max_dist), &max_dist, sizeof(double),0,hipMemcpyHostToDevice,streams[5]));
    gpuErrchk( hipMemcpyToSymbolAsync(HIP_SYMBOL( d_totalVuln), &totalVuln, sizeof(int),0,hipMemcpyHostToDevice,streams[6]));


    size_t h_pitchBytes = n_colegios * sizeof(double);
    hipMemcpy2DAsync(d_distMat,
                 pitch,
                 matrestest,
                 h_pitchBytes,
                 n_colegios * sizeof(double),
                 n_students,
                 hipMemcpyHostToDevice,
                 streams[3]);


    ///////////////////////////////////////////////////
    /// Inicializa las distribuciónes
    ///////////////////////////////////////////////////

    dist = std::uniform_int_distribution<int>(0, n_students-1);
    dist2 = std::uniform_int_distribution<int>(0, n_colegios-1);

    ///////////////////////////////////////////////////
    /// Contador de tiempo de ejecución en cuda
    ///////////////////////////////////////////////////

    hipEvent_t start_cuda;
    hipEvent_t stop_cuda;
    hipEventCreate(&start_cuda);
    hipEventCreate(&stop_cuda);
    // float elapsedTime;
    // double timeCuda = 0.0;
    ///////////////////////////////////////////////////
    /// Inicio el contador de tiempo antes de iniciar el algortimo
    ///////////////////////////////////////////////////
    //int vef_count = 0;


    hipMemcpyAsync(d_currentSolution, currentSolution, n_students * sizeof(int), hipMemcpyHostToDevice,streams[2]);
    hipMemcpyAsync(d_previousSolution, currentSolution, n_students * sizeof(int), hipMemcpyHostToDevice,streams[3]);
    hipMemcpyAsync(d_bestSolution, currentSolution, n_students * sizeof(int), hipMemcpyHostToDevice,streams[4]);
    hipMemcpyAsync(d_aluxcol, aluxcol, n_colegios * sizeof(int), hipMemcpyHostToDevice,streams[5]);
    hipMemcpyAsync(d_previousAluxcol, aluxcol, n_colegios * sizeof(int), hipMemcpyHostToDevice,streams[6]);
    hipMemcpyAsync(d_aluVulxCol, aluVulxCol, n_colegios * sizeof(int), hipMemcpyHostToDevice,streams[7]);
    hipMemcpyAsync(d_previousAluVulxCol, aluVulxCol, n_colegios * sizeof(int), hipMemcpyHostToDevice,streams[8]);
    hipMemcpyAsync(d_currentVars, currentVars, 3 * sizeof(double), hipMemcpyHostToDevice,streams[9]);
    hipMemcpyAsync(d_previousVars, currentVars, 3 * sizeof(double), hipMemcpyHostToDevice,streams[0]);
    hipMemcpyAsync(d_bestVars, currentVars, 3 * sizeof(double), hipMemcpyHostToDevice,streams[1]);
    hipMemcpyAsync(d_alumnosSep, alumnosSep, n_students * sizeof(int), hipMemcpyHostToDevice,streams[2]);
    hipMemcpyAsync(d_cupoArray, cupoArray, n_colegios * sizeof(int), hipMemcpyHostToDevice,streams[3]);

    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    
    if (errSync != hipSuccess) 
        printf("0 Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("0 Async kernel error: %s\n", hipGetErrorString(errAsync));
    ///////////////////////////// Incorporar para acceder mas rapido al costCurrentSolution
    //int deviceId;
    //hipGetDevice(&deviceId);                                         // The ID of the currently active GPU device.
    //hipMemPrefetchAsync(pointerToSomeUMData, size, deviceId); 

    CoolingScheme cooling = CoolingScheme(&temp, coolingRate);
    //Reheating reheating = Reheating(&temp, &k_reheating, &n_reheating);

    while(cooling.getTemp() > min_temp){

        copyMemSolution<<<numberOfBlocks,threadsPerBlock,0,streams[0]>>>(d_currentSolution, d_previousSolution,n_students);
        copyMemCol<<<numberOfBlocks,threadsPerBlock,0,streams[1]>>>(d_aluxcol, d_previousAluxcol,n_colegios);
        copyMemCol<<<numberOfBlocks,threadsPerBlock,0,streams[2]>>>(d_aluVulxCol, d_previousAluVulxCol,n_colegios);
        copyVars<<<1,3,0,streams[3]>>>(d_currentVars, d_previousVars);
        errSync  = hipGetLastError();
        errAsync = hipDeviceSynchronize();
        if (errSync != hipSuccess) 
        printf("1 Sync kernel error: %s\n", hipGetErrorString(errSync));
        if (errAsync != hipSuccess)
        printf("1 Async kernel error: %s\n", hipGetErrorString(errAsync));
        //for (int i = 0; i < NUM_STREAMS; ++i) { hipStreamSynchronize(streams[i]); }

        /*
        memcpy(currentSolution,previousSolution,sizeof(int)*n_students);
        memcpy(aluxcol,previousAluxCol,sizeof(int)*n_colegios);
        memcpy(aluVulxCol,previousAluVulxCol,sizeof(int)*n_colegios);
        memcpy(currentVars,previousVars,sizeof(double)*3);
        */

        ///////////////////////////////////////////////////
        ///  Selecciona aleatoria mente a los alumnos
        ///////////////////////////////////////////////////

        shuffle(shuffle_student, max_changes_students, dist);
        shuffle(shuffle_colegios, max_changes_school, dist2);
        ///////////////////////////////////////////////////
        /// Actualiza la memoria en CUDA
        ///////////////////////////////////////////////////
        /*
        hipMemcpy(d_currentSolution, currentSolution, n_students * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_aluxcol, aluxcol, n_colegios * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_aluVulxCol, aluVulxCol, n_colegios * sizeof(int), hipMemcpyHostToDevice);
        */



        ///////////////////////////////////////////////////
        ///  Envia datos a GPU
        ///////////////////////////////////////////////////

        
        hipMemcpyAsync(d_shuffle_students, shuffle_student, max_changes_students* sizeof(int), hipMemcpyHostToDevice,streams[0]);
        hipMemcpyAsync(d_shuffle_colegios, shuffle_colegios, max_changes_school * sizeof(int), hipMemcpyHostToDevice,streams[1]);
        errSync  = hipGetLastError();
        errAsync = hipDeviceSynchronize();
        if (errSync != hipSuccess) 
        printf("2 Sync kernel error: %s\n", hipGetErrorString(errSync));
        if (errAsync != hipSuccess)
        printf("2 Async kernel error: %s\n", hipGetErrorString(errAsync));

        ///////////////////////////////////////////////////
        ///  Ejecuta los kernel
        //////////////////////////////////////////////////
        //cout << (n_block/nWarp+1) << endl;
        newSolution_kernel<<<n_block,n_thread,
        (n_thread/nWarp+1) * sizeof(double)+ (n_thread/nWarp+1)* sizeof(int) + (n_thread/nWarp+1)* sizeof(int)>>>(
                        d_array_current_Solution,
                                d_array_current_Solution_alu,
                                d_array_current_Solution_col,
                                d_cupoArray,
                                d_alumnosSep,
                                d_aluxcol,
                                d_aluVulxCol,
                                d_currentSolution,
                                d_distMat,
                                d_shuffle_students,
                                d_shuffle_colegios,
                                d_currentVars,
                                pitch);
        errSync  = hipGetLastError();
        errAsync = hipDeviceSynchronize();
        if (errSync != hipSuccess) 
        printf("3 Sync kernel error: %s\n", hipGetErrorString(errSync));
        if (errAsync != hipSuccess)
        printf("3 Async kernel error: %s\n", hipGetErrorString(errAsync));
        reduce_block_kernel<<<1,n_block,
        (n_block/nWarp+1)* sizeof(double)+ (n_block/nWarp+1)* sizeof(int)+ (n_block/nWarp+1)* sizeof(int)>>>(d_array_current_Solution,
                d_array_current_Solution_alu,
                d_array_current_Solution_col);
        errSync  = hipGetLastError();
        errAsync = hipDeviceSynchronize();
        if (errSync != hipSuccess) 
        printf("4 Sync kernel error: %s\n", hipGetErrorString(errSync));
        if (errAsync != hipSuccess)
        printf("4 Async kernel error: %s\n", hipGetErrorString(errAsync));

        /********************************
        /* Metodo Nuevo
        */
        hipMemcpy(&costCurrentSolution,&d_array_current_Solution[0], sizeof(double),hipMemcpyDeviceToHost);
        errSync  = hipGetLastError();
        errAsync = hipDeviceSynchronize();
        if (errSync != hipSuccess) 
        printf("5 Sync kernel error: %s\n", hipGetErrorString(errSync));
        if (errAsync != hipSuccess)
        printf("5 Async kernel error: %s\n", hipGetErrorString(errAsync));

        if(costCurrentSolution >= costPreviousSolution){
            if(metropolisAC1(costPreviousSolution,costCurrentSolution)==1){
                selectThread = dist(mt);
                selectBlock = dist2(mt);
                hipMemcpy(&d_array_current_Solution_alu[0],&selectThread, sizeof(int),hipMemcpyHostToDevice);
                hipMemcpy(d_array_current_Solution_col,&selectBlock, sizeof(int),hipMemcpyHostToDevice);
                errSync  = hipGetLastError();
                errAsync = hipDeviceSynchronize();
                if (errSync != hipSuccess) 
                printf("6 Sync kernel error: %s\n", hipGetErrorString(errSync));
                if (errAsync != hipSuccess)
                printf("6 Async kernel error: %s\n", hipGetErrorString(errAsync));
                //cout << "son iguales" << endl;
            }
        }

        /* best
        if(costCurrentSolution >= costPreviousSolution){
            if(costCurrentSolution > costPreviousSolution){
                selectThread = dist(mt);
                selectBlock = dist2(mt);
                hipMemcpy(&d_array_current_Solution_alu[0],&selectThread, sizeof(int),hipMemcpyHostToDevice);
                hipMemcpy(d_array_current_Solution_col,&selectBlock, sizeof(int),hipMemcpyHostToDevice);
                errSync  = hipGetLastError();
                errAsync = hipDeviceSynchronize();
                if (errSync != hipSuccess) 
                printf("6 Sync kernel error: %s\n", hipGetErrorString(errSync));
                if (errAsync != hipSuccess)
                printf("6 Async kernel error: %s\n", hipGetErrorString(errAsync));
                //cout << "son iguales" << endl;
                }
            else{
                if(metropolisAC1(costPreviousSolution,costCurrentSolution)==1){
                    reduce_block_max<<<1,n_block,
                    (n_block/nWarp+1)* sizeof(double)+ (n_block/nWarp+1)* sizeof(int)+ (n_block/nWarp+1)* sizeof(int)>>>(d_array_current_Solution,
                    d_array_current_Solution_alu,
                    d_array_current_Solution_col);
                    errSync  = hipGetLastError();
                    errAsync = hipDeviceSynchronize();
                    if (errSync != hipSuccess) 
                    printf("6 Sync kernel error: %s\n", hipGetErrorString(errSync));
                    if (errAsync != hipSuccess)
                    printf("6 Async kernel error: %s\n", hipGetErrorString(errAsync));
                }
            }
            
        }
        */
        calculateSolution<<<1,1>>>(d_array_current_Solution,
            d_array_current_Solution_alu,
            d_array_current_Solution_col,
            d_cupoArray,
            d_alumnosSep,
            d_aluxcol,
            d_aluVulxCol,
            d_currentSolution,
            d_distMat,
            pitch,
            d_currentVars,
            d_costCurrentSolution);
        hipMemcpy(&costCurrentSolution,&d_array_current_Solution[0], sizeof(double),hipMemcpyDeviceToHost);
        //hipMemcpyAsync(&selectThread,&d_array_current_Solution_alu[0], sizeof(int),hipMemcpyDeviceToHost,streams[1]);
        //hipMemcpyAsync(&selectBlock,d_array_current_Solution_col, sizeof(int),hipMemcpyDeviceToHost,streams[2]);
        errSync  = hipGetLastError();
        errAsync = hipDeviceSynchronize();
        if (errSync != hipSuccess) 
        printf("8 Sync kernel error: %s\n", hipGetErrorString(errSync));
        if (errAsync != hipSuccess)
        printf("8 Async kernel error: %s\n", hipGetErrorString(errAsync));

        
        /********************************
        // Metodo antiguo
        */
        //cout << endl;
        /*
        calculateSolution<<<1,1>>>(d_array_current_Solution,
                    d_array_current_Solution_alu,
                    d_array_current_Solution_col,
                    d_cupoArray,
                    d_alumnosSep,
                    d_aluxcol,
                    d_aluVulxCol,
                    d_currentSolution,
                    d_distMat,
                    pitch,
                    d_currentVars,
                    d_costCurrentSolution);
        hipMemcpy(&costCurrentSolution,&d_array_current_Solution[0], sizeof(double),hipMemcpyDeviceToHost);
        //hipMemcpyAsync(&selectThread,&d_array_current_Solution_alu[0], sizeof(int),hipMemcpyDeviceToHost,streams[1]);
        //hipMemcpyAsync(&selectBlock,d_array_current_Solution_col, sizeof(int),hipMemcpyDeviceToHost,streams[2]);
        hipDeviceSynchronize();
        */
        /*
        if(costCurrentSolution > costBestSolution){
            if(metropolisAC1(costPreviousSolution,costCurrentSolution)==1){
            reduce_block_max<<<1,n_block,
            (n_block/nWarp+1)* sizeof(double)+ (n_block/nWarp+1)* sizeof(int)+ (n_block/nWarp+1)* sizeof(int)>>>(d_array_current_Solution,
            d_array_current_Solution_alu,
            d_array_current_Solution_col);
            calculateSolution<<<1,1>>>(d_array_current_Solution,
                d_array_current_Solution_alu,
                d_array_current_Solution_col,
                d_cupoArray,
                d_alumnosSep,
                d_aluxcol,
                d_aluVulxCol,
                d_currentSolution,
                d_distMat,
                pitch,
                d_currentVars,
                d_costCurrentSolution);
            hipMemcpy(&costCurrentSolution,&d_array_current_Solution[0], sizeof(double),hipMemcpyDeviceToHost);
            //hipMemcpyAsync(&selectThread,&d_array_current_Solution_alu[0], sizeof(int),hipMemcpyDeviceToHost,streams[1]);
            //hipMemcpyAsync(&selectBlock,d_array_current_Solution_col, sizeof(int),hipMemcpyDeviceToHost,streams[2]);
            hipDeviceSynchronize();
            }
        }*/
        
        //exit(0);
        ///////////////////////////////////////////////////
        ///  Actualizo datos basicos
        ///////////////////////////////////////////////////
    
        /*
        vector_historyAsign.push_back(currentSolution[shuffle_student[selectBlock]]);           
        aluxcol[currentSolution[shuffle_student[selectBlock]]]-=1; ///
        aluVulxCol[currentSolution[shuffle_student[selectBlock]]]-=alumnosSep[shuffle_student[selectBlock]]; ///
        aluxcol[shuffle_colegios[selectThread]]+=1; ///
        aluVulxCol[shuffle_colegios[selectThread]]+=alumnosSep[shuffle_student[selectBlock]]; ///
        currentSolution[shuffle_student[selectBlock]] = shuffle_colegios[selectThread]; ///
        */

        ///////////////////////////////////////////////////
        /// Salida en caso de error
        ///////////////////////////////////////////////////
        //std::cout << costCurrentSolution << "\n";
        //std::cout << selectThread << "\n";
        //std::cout << selectBlock << "\n";
        
        if(costCurrentSolution<0.00 || isnan(costCurrentSolution)){
            std::cout << shuffle_colegios[selectThread] << "\n";
            std::cout << shuffle_student[selectBlock] << "\n";
            std::cout << "distancia: " << meanDist(currentSolution,distMat) << "\n";
            std::cout << "Segregación: " << S(currentSolution,alumnosSep, totalVuln) << "\n";
            std::cout << "CostoCupo: " << costCupo(currentSolution,cupoArray) << "\n";
            std::cout << costCurrentSolution;
            exit(1);
        }
        
        
        
        if(costCurrentSolution < costBestSolution){
            
            copyMemSolution<<<numberOfBlocks,threadsPerBlock,0,streams[0]>>>(d_bestSolution, d_currentSolution,n_students);
            copyMemSolution<<<numberOfBlocks,threadsPerBlock,0,streams[1]>>>(d_previousSolution, d_currentSolution,n_students);
            copyMemCol<<<numberOfBlocks,threadsPerBlock,0,streams[2]>>>(d_previousAluxcol, d_aluxcol,n_colegios);
            copyMemCol<<<numberOfBlocks,threadsPerBlock,0,streams[3]>>>(d_previousAluVulxCol, d_aluVulxCol,n_colegios);
            copyVars<<<1,3,0,streams[4]>>>(d_previousVars, d_currentVars);
            copyVars<<<1,3,0,streams[5]>>>(d_bestVars, d_currentVars);
            copyCost<<<1,1,0,streams[6]>>>(d_costBestSolution,d_costCurrentSolution);
            copyCost<<<1,1,0,streams[7]>>>(d_costPreviousSolution,d_costCurrentSolution);
            //for (int i = 0; i < NUM_STREAMS; ++i) { hipStreamSynchronize(streams[i]); }
            errSync  = hipGetLastError();
            errAsync = hipDeviceSynchronize();
            if (errSync != hipSuccess) 
            printf("9 Sync kernel error: %s\n", hipGetErrorString(errSync));
            if (errAsync != hipSuccess)
            printf("9 Async kernel error: %s\n", hipGetErrorString(errAsync));
            /*
            memcpy(bestSolution,currentSolution,sizeof(int)*n_students);
            memcpy(previousSolution,currentSolution,sizeof(int)*n_students);
            memcpy(previousAluxCol,aluxcol,sizeof(int)*n_colegios);
            memcpy(previousAluVulxCol,aluVulxCol,sizeof(int)*n_colegios);
            memcpy(previousVars,currentVars,sizeof(double)*3);
            memcpy(bestVars,currentVars,sizeof(double)*3);
            */
            
            costBestSolution = costCurrentSolution;
            costPreviousSolution = costCurrentSolution;
            //cout << costBestSolution << "| |" << temp << "| |" << count<< endl;
            /*
            vector_costCurrentSolution.push_back(costCurrentSolution);
            vector_meanDist.push_back(meanDist(currentSolution,distMat));
            vector_segregation.push_back(S(currentSolution, alumnosSep, totalVuln));
            vector_costoCupo.push_back(costCupo(currentSolution,cupoArray));
            vector_temp.push_back(temp);
            vector_count.push_back(count);
            */
            c_accepta++;
            count_rechaso = 0;
        }
        // En el caso que el la solución actual sea mas alta intenta aceptar una peor solución en base
        // a la función acepta
        else{
            if(metropolisAC1(costPreviousSolution,costCurrentSolution) == 1) {

                copyMemSolution<<<numberOfBlocks,threadsPerBlock,0,streams[0]>>>(d_previousSolution, d_currentSolution,n_students);
                copyMemCol<<<numberOfBlocks,threadsPerBlock,0,streams[1]>>>(d_previousAluxcol, d_aluxcol,n_colegios);
                copyMemCol<<<numberOfBlocks,threadsPerBlock,0,streams[2]>>>(d_previousAluVulxCol, d_aluVulxCol,n_colegios);
                copyVars<<<1,3,0,streams[3]>>>(d_previousVars, d_currentVars);
                copyCost<<<1,1,0,streams[4]>>>(d_costPreviousSolution,d_costCurrentSolution);
                //for (int i = 0; i < NUM_STREAMS; ++i) { hipStreamSynchronize(streams[i]); }
                errSync = hipGetLastError();
                errAsync = hipDeviceSynchronize();
                if (errSync != hipSuccess) 
                printf("10 Sync kernel error: %s\n", hipGetErrorString(errSync));
                if (errAsync != hipSuccess)
                printf("10 Async kernel error: %s\n", hipGetErrorString(errAsync));

                /*
                memcpy(previousSolution,currentSolution,sizeof(int)*n_students);
                memcpy(previousAluxCol,aluxcol,sizeof(int)*n_colegios);
                memcpy(previousAluVulxCol,aluVulxCol,sizeof(int)*n_colegios);
                memcpy(previousVars,currentVars,sizeof(double)*3);
                */
                costPreviousSolution = costCurrentSolution;

                count_rechaso = 0;
                c_accepta++;
            }
            else{
                count_rechaso++;
                
            }
        }

        if(temperatureTL7(c_cooling_temperature, c_accepta, len1, len2, n_colegios, count)){
        //if(temperatureTL8(temp, c_cooling_temperature, count_trials, len1, len2, coolingRate)){
        //if(temperatureTL9(temp, c_cooling_temperature, count_trials, len3, len4, coolingRate)){
        //if(temperatureTL11(temp, c_cooling_temperature, count_trials, len3, len4, coolingRate)){
            cooling.CS2();
            cout << cooling.getTemp() << "\n";
        }

        //reheating.TR11(temp, k_reheating, n_reheating, count_rechaso);
        //reheating.TR12(temp, k_reheating, n_reheating, count);
        //reheating.TR13(temp, k_reheating, n_reheating, c_cooling_temperature);
        //reheating.TR14(temp, k_reheating, k_reheating_init, n_reheating, count_rechaso, e_const);
        
        
        
        ///////////////////////////////////////////////////
        /// History
        ///////////////////////////////////////////////////
        /*
        vector_historyCostSolution.push_back(costCurrentSolution);
        vector_historyTemp.push_back(temp);
        vector_historymeanDist.push_back(meanDist(currentSolution,distMat));
        vector_historymeanDistNorm.push_back(meanDist(currentSolution,distMat)/max_dist);
        vector_historySegregation.push_back(S(currentSolution, alumnosSep, totalVuln));
        vector_historycostoCupo.push_back(costCupo(currentSolution,cupoArray));
        if(count_rechaso==0){
            vector_historyAcceptSolution.push_back(1);
        }
        else{
            vector_historyAcceptSolution.push_back(0);
        }
        vector_historyMove.push_back(std::tuple<int,int>(shuffle_colegios[selectThread],shuffle_student[selectBlock]));     
        */
        
        //cout << costCurrentSolution << costPreviousSolution << "| |" << temp << "| |" << count<< endl;
        errSync  = hipGetLastError();
        errAsync = hipDeviceSynchronize();
        if (errSync != hipSuccess) 
        printf("6 Sync kernel error: %s\n", hipGetErrorString(errSync));
        if (errAsync != hipSuccess)
        printf("6 Async kernel error: %s\n", hipGetErrorString(errAsync));
        count_trials++;
        count++;
    }
    hipMemcpyAsync(bestSolution, d_bestSolution, n_students * sizeof(int), hipMemcpyDeviceToHost,streams[0]);
    hipMemcpyAsync(previousSolution, d_previousSolution, n_students * sizeof(int), hipMemcpyDeviceToHost,streams[1]);
    ///////////////////////////////////////////////////
    /// Obtiene el tiempo de ejecución
    ///////////////////////////////////////////////////
    auto end = std::chrono::high_resolution_clock::now();
    double time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_taken *= 1e-9;

    for(x=0;x<n_students;x++){
        info_graficos_bestSolution << bestSolution[x] << ",";
    }
    

    for(x=0; x<vector_count.size(); x++){
        info_graficos << vector_count.at(x) << "," 
                    << vector_meanDist.at(x)/max_dist << "," // Distancia promedio recorrida por los estudiantes normalizada
                    << vector_meanDist.at(x) << "," 
                    << vector_segregation.at(x) << "," 
                    << vector_costoCupo.at(x) << "," 
                    << vector_costCurrentSolution.at(x) << "," 
                    << fixed << vector_temp.at(x) << setprecision(13) << "\n";
    }




    ///////////////////////////////////////////////////
    /// Almacenamiento de datos
    ///////////////////////////////////////////////////
    cout.precision(dbl::max_digits10);
    cout << "--------------- Resultado Final ----------------" << "\n";
    cout << "Numero de Ciclos " << count << "\n";
    cout << "Costo de la solución previa: " << costPreviousSolution << "\n";
    cout << "Costo de la mejor solución: " << costBestSolution << "\n";
    cout << "Costo de la solución actual: " << costCurrentSolution << "\n";
    cout << "Tiempo de ejecución de SA: " << time_taken << "\n";
    cout << "distancia: " << meanDist(bestSolution,distMat) << "\n";
    cout << "Segregación: " << S(bestSolution, alumnosSep, totalVuln) << "\n";
    cout << "CostoCupo: " << costCupo(bestSolution,cupoArray) << "\n";

    cout << "Cal costo " << calCosto(bestSolution, distMat, ptr_alpha, alumnosSep, totalVuln, cupoArray) << endl;
    cout << "Costo de: " << costBestSolution << "\n";


    //cout << fixed << setprecision(70) << endl;
    cout << sumDist(bestSolution,distMat) << "\n";
    cout << bestVars[0] << endl;
    cout << sumS(bestSolution, alumnosSep, totalVuln) << "\n";
    cout << bestVars[1] << endl;
    cout << sumCostCupo(bestSolution,cupoArray) << "\n";
    cout << bestVars[2] << endl;
    cout << "Tiempo de ejecución de SA get_result: " << vector_time1 << "\n";

    cout << "--------------- Finalizo con exito ----------------" << "\n";


    info << "--------------- Resultado Final ----------------" << "\n";
    info << "Numero de Ciclos " << count << "\n";
    info << "Costo de la solución previa: " << costPreviousSolution << "\n";
    info << "Costo de la mejor solución: " << costBestSolution << "\n";
    info << "Costo de la solución actual: " << costCurrentSolution << "\n";
    info << "Tiempo de ejecución de SA: " << time_taken << "\n";
    info << "distancia: " << meanDist(bestSolution,distMat) << "\n";
    info << "Segregación: " << S(bestSolution, alumnosSep, totalVuln) << "\n";
    info << "CostoCupo: " << costCupo(bestSolution,cupoArray) << "\n";
    info << "--------------- Finalizo con exito ----------------" << "\n";


    info_test << fixed << time_taken << setprecision(9) << "," 
            << costBestSolution << "," 
            << meanDist(bestSolution,distMat)/max_dist 
            << "," << meanDist(bestSolution,distMat) 
            << "," << S(bestSolution, alumnosSep, totalVuln) 
            << "," << costCupo(bestSolution,cupoArray) 
            << "," << count 
            << "," << fixed << temp_init << setprecision(13) 
            << "," << fixed << cooling.getTemp() << setprecision(13) 
            << "," << min_temp 
            << "," << seed
            << "," << alpha1 
            << "," << alpha2 
            << "," << alpha3 
            << "," << alpha[0]
            << "," << alpha[1]
            << "," << alpha[2]
            << "," << coolingRate 
            << "," << k_reheating_init 
            << "," << e_const
            << "," << n_reheating
            << "," << len1_init
            << "," << len2_init
            << "," << len3_init
            << "," << len4_init
            << "," << len1
            << "," << len2
            << "," << len3
            << "," << len4
            << "," << Th
            << "," << n_block 
            << "," << n_thread 
            << ","<< name_exp << "\n";

    info_graficos_bestSolution.close();
    cout << ".";
    info_graficos.close();
    cout << ".";
    info_test.close();
    info.close();
    cout << ".\n";
    cout << " Archivos Guardado" << "\n";



    for (int i = 0; i < NUM_STREAMS; ++i) { hipStreamDestroy(streams[i]); }
    hipFree(d_array_current_Solution);
    hipFree(d_costCurrentSolution);
    hipFree(d_costBestSolution);
    hipFree(d_costPreviousSolution);
    hipFree(d_currentVars);
    hipFree(d_bestVars);
    hipFree(d_previousVars);
    hipFree(d_array_current_Solution_alu);
    hipFree(d_array_current_Solution_col);
    hipFree(d_shuffle_colegios);
    hipFree(d_shuffle_students);
    hipFree(d_aluxcol);
    hipFree(d_previousAluxcol);
    hipFree(d_aluVulxCol);
    hipFree(d_previousAluVulxCol);
    hipFree(d_currentSolution);
    hipFree(d_bestSolution);
    hipFree(d_previousSolution);
    hipFree(d_alumnosSep);
    hipFree(d_cupoArray);
    hipFree(d_distMat);
    hipFree(d_alpha);

    hipEventDestroy(start_cuda);
    hipEventDestroy(stop_cuda);

    return (costBestSolution);

}

///////////////////////////////////////////////////
///////////////////////////////////////////////////


///////////////////////////////////////////////////
/// Calcula el costo
///////////////////////////////////////////////////
double calCosto(int currentSolution[], double **distMat, const double ptr_alpha[], int alumnosSep[], int totalVuln, int cupoArray[]){
    double var1 = meanDist(currentSolution,distMat)/max_dist;
    //cout << "distancia: " << var1 << "\n";
    double var2 = S(currentSolution, alumnosSep, totalVuln);
    //cout << "Segregación: " << var2 << "\n";
    double var3 = costCupo(currentSolution,cupoArray);
    //cout << "CostoCupo: " << var3 << "\n";
    return (double)((ptr_alpha[0]*var1)+(ptr_alpha[1]*var2)+(ptr_alpha[2]*var3));
}

///////////////////////////////////////////////////
/// Distancia promedio que recorreran los estudiantes
///////////////////////////////////////////////////
double meanDist(const int currentSolution[], double  **distMat){
    double sumDist=0.0;
    for(int i=0;i<n_students;i++){
        sumDist+=round_n(distMat[i][currentSolution[i]]); // distMat[estudiante][escuela]
    }
    //cout << "meanDist: " << sumDist << endl;
    //cout << "Numero de estudiantes: " << n_students << "  |  Suma de distancias:" << sumDist << "\n";
    return sumDist/n_students;
}

double sumDist(const int currentSolution[], double  **distMat){
    double sumDist=0.0;
    for(int i=0;i<n_students;i++){
        sumDist+=round_n(distMat[i][currentSolution[i]]); // distMat[estudiante][escuela]
    }
    //cout << "sumDist: " << sumDist << endl;
    //cout << "Numero de estudiantes: " << n_students << "  |  Suma de distancias:" << sumDist << "\n";
    return sumDist;
}


///////////////////////////////////////////////////
/// Calcula segregación por duncan
///////////////////////////////////////////////////

double S(const int currentSolution[],const int alumnosSep[], int totalVuln){
    double totalSesc = 0.0;
    int aluVulCol =0;
    int aluNoVulCol = 0;
    for(int n=0; n<n_colegios;n++){
        aluVulCol = 0;
        aluNoVulCol = 0;
        for (int a = 0; a < n_students; a++){
            if(currentSolution[a] == n){
                aluNoVulCol++;
                aluVulCol+=alumnosSep[a];
            }
        }
        if(aluNoVulCol>0){
            aluNoVulCol =aluNoVulCol - aluVulCol;
            totalSesc+=round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
        }
    }
    return totalSesc/2.0;
}

double sumS(const int currentSolution[],const int alumnosSep[], int totalVuln){
    double totalSesc = 0.0;
    int aluVulCol =0;
    int aluNoVulCol = 0;
    for(int n=0; n<n_colegios;n++){
        aluVulCol = 0;
        aluNoVulCol = 0;
        for (int a = 0; a < n_students; a++){
            if(currentSolution[a] == n){
                aluNoVulCol++;
                aluVulCol+=alumnosSep[a];
            }
        }
        if(aluNoVulCol>0){
            aluNoVulCol =aluNoVulCol - aluVulCol;
            totalSesc+=round_n(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
        }
    }
    return totalSesc;
}


///////////////////////////////////////////////////
/// Calcula el costo de tener los estudiantes en las escuelas
///////////////////////////////////////////////////

double costCupo(int currentSolution[],int cupoArray[]){
    double totalcostCupo = 0.0;
    int totalAluCol = 0;
    // double a = 0.0;
    for(int j=0;j<n_colegios;j++){
        totalAluCol = 0;
        for(int i=0; i<n_students; i++){
            if(currentSolution[i]==j){
                totalAluCol++;
            }
        }
        totalcostCupo+=round_n((double)totalAluCol*fabs(((double)cupoArray[j]-totalAluCol)/pow(((double)cupoArray[j]/2),2)));
    }
    return totalcostCupo/n_colegios;
}



double sumCostCupo(int currentSolution[],int cupoArray[]){
    double totalcostCupo = 0.0;
    int totalAluCol = 0;
    for(int j=0;j<n_colegios;j++){
        totalAluCol = 0;
        for(int i=0; i<n_students; i++){
            if(currentSolution[i]==j){
                totalAluCol++;
            }
        }
        totalcostCupo+= round_n((double)totalAluCol*fabs(((double)cupoArray[j]-totalAluCol)/pow(((double)cupoArray[j]/2),2)));
    }
    return totalcostCupo;
}
///////////////////////////////////////////////////
/// Genera una nueva solución en donde asigna a un estudiante a una escuela
/// aleatoriamente
///////////////////////////////////////////////////

void newSolution(int currentSolution[],const int previousSolution[]){
    random_device rd;
    mt19937 mt(rd());
    uniform_int_distribution<int> dist(0, n_students);
    random_device rd2;
    mt19937 mt2(rd2());
    uniform_int_distribution<int> dist2(0, n_colegios);
    int selectStudent=dist(mt);
    int selectSchool = dist2(mt2);
    for(int x=0; x<n_students; x++){
        if(x == selectStudent) {
            currentSolution[x] = selectSchool;
        }
        else {
            currentSolution[x] = previousSolution[x];
        }
    }

}




///////////////////////////////////////////////////
/// Asigna a las soluciones la escuela actual Solo se utiliza al inicio
///////////////////////////////////////////////////
void assignSchoolToArray(int previousSolution[], int bestSolution[], int currentSolution[], Info_colegio *ptr_colegios, Info_alu *ptr_students, int cupoArray[]){
    Info_alu *ptr_aux = ptr_students;
    for(int x=0;x < n_colegios;x++){
        for(int y=0; y < n_students; y++){
            if(ptr_colegios->rbd == ptr_students->rbd){
                previousSolution[y] = x;
                bestSolution[y] = x;
                currentSolution[y] = x;
            }
            ptr_students++;

        }
        /*
         * cupoArray sera un arreglo que por indice es la escuela y su valor sera el cupo que posee esa escuela
         * se asume que las escuelas pueden tener sobre cupo.
         */

        cupoArray[x] = ptr_colegios->num_alu+ ((int)((ptr_colegios->num_alu*10)/100));
        ptr_students = ptr_aux;
        ptr_colegios++;
    }
}
///////////////////////////////////////////////////
/// Crea una matriz de distancia donde x es el estudiante, y es la escuela
///////////////////////////////////////////////////
void calcDist(Info_colegio *ptr_colegios, Info_alu *ptr_students, double **distMat){
    Info_colegio *ptr_aux = ptr_colegios;
    for(int x=0;x < n_students ;x++){
        for(int y=0; y < n_colegios; y++){
            distMat[x][y] = sqrt( pow((ptr_students->latitude - ptr_colegios->latitude),2)+pow((ptr_students->longitude - ptr_colegios->longitude),2))/1000;
            ptr_colegios++;

        }
        ptr_colegios = ptr_aux;
        ptr_students++;
    }
}

///////////////////////////////////////////////////
/// newSolution_v2, tiene como entrada la información de los estado actual de la solución, y alcula de inmediato la
/// distancia promedio, el costocupo y segregación total.
///////////////////////////////////////////////////
double newSolution_v2(int n_students,int n_colegios,int totalVuln,int aluxcol[],int aluVulxCol[],int cupoArray[],double **distMat, int currentSolution[], const double ptr_alpha[]){
    double sumDist=0;
    // double mean=0.0;
    double totalcostCupo = 0.0;
    double totalSesc = 0.0;
    int aluVulCol, aluNoVulCol,totalAluCol;
    for(int i=0;i<n_students;i++){
        sumDist+=distMat[i][currentSolution[i]]; // distMat[estudiante][escuela]
    }
    for(int n=0; n<n_colegios; n++) {
        totalAluCol = aluxcol[n];
        aluVulCol = aluVulxCol[n];
        aluNoVulCol =totalAluCol - aluVulCol;
        // Calcula el costo cupo
        totalcostCupo+=totalAluCol*fabs((cupoArray[n]-totalAluCol)/pow(((double)cupoArray[n]/2),2));
        // Calcula el total sesc
        totalSesc+=((double)1/2)*fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln)));
    }
    totalcostCupo = totalcostCupo/n_colegios;
    double var1 = (sumDist/(double)n_students)/max_dist;
    //cout << var1 << "\n";
    double var2 = totalSesc;
    //cout << var2 << "\n";
    double var3 = totalcostCupo;
    //cout << var3 << "\n";
    return (double)((ptr_alpha[0]*var1)+(ptr_alpha[1]*var2)+(ptr_alpha[2]*var3));
}

void shuffle(int *values, const int max_change, uniform_int_distribution<int> distri) {
    int randvalue1,randvalue2,tem_value;
    for (int i = 0; i<max_change; i++) {
        randvalue1 = distri(mt);
        randvalue2 = i;
        tem_value = values[randvalue1];
        values[randvalue1] = values[randvalue2];
        values[randvalue2] = tem_value;
    }
}

void getDataSchool(vector<Info_colegio> &colegios){
    string line_colegios;
    ifstream info_school("colegios_utm.txt"); // concatenar
    int cx = 0;
    while (getline(info_school, line_colegios)) {
        stringstream linestream(line_colegios);
        string data;
        colegios.push_back(Info_colegio());
        getline(linestream, data, ',');
        colegios[cx].rbd = stoi(data);
        getline(linestream, data, ',');
        colegios[cx].latitude = stod(data);
        getline(linestream, data, ',');
        colegios[cx].longitude = stod(data);
        getline(linestream, data, ',');
        colegios[cx].num_alu = stoi(data);
        getline(linestream, data, ',');
        colegios[cx].prioritario = stoi(data);
        cx++;
    }
    info_school.close();
}

void getDataStudents(vector<Info_alu> &students, int &totalVuln)
{
    string line_student;
    ifstream info_student("alumnos_utm.txt"); // concatenar
    int cx = 0;
    while (getline(info_student, line_student)) {
        stringstream linestream(line_student);
        string data;
        students.push_back(Info_alu());
        getline(linestream, data, ',');
        students[cx].rbd = stoi(data);
        getline(linestream, data, ',');
        students[cx].latitude = stod(data);
        getline(linestream, data, ',');
        students[cx].longitude = stod(data);
        getline(linestream, data, ',');
        students[cx].sep = stoi(data);
        if (students[cx].sep == 1) {
            totalVuln++;
        }
        cx++;

    }
    info_student.close();
}

////////////////////////////////////////////////
////// Obtiene la maxima distancia que un estudiante podria llegar a recorrer
///////////////////////////////////////////////////
double getMaxDistance(double **distMat){
    double max = 0;
    for(int i=0;i<n_students;i++){
        for(int x=0;x<n_colegios;x++){
            if(distMat[i][x]>max){
                max = distMat[i][x];
            }
        }
    }
    return max;
}

///////////////////////////////////////////////////
/// Calcula el valor de los alpha
///////////////////////////////////////////////////
void normalizedAlpha(double alpha[3])
{
    double sumaAlpha = 0.0;
    for(int x=0; x<3; x++){
        sumaAlpha +=alpha[x];
    }
    for(int x=0; x<3; x++){
        alpha[x]= alpha[x]/(double)sumaAlpha;
    }
}


///////////////////////////////////////////////////
/// Asigna Información de las escuelas a best, previus y current soluciones
///////////////////////////////////////////////////
void initializeArray(int *aluxcol, int *previousAluxCol, int *bestAluxCol, int *aluVulxCol, int *previousAluVulxCol, int *bestAluVulxCol, int *alumnosSep, vector<Info_alu> &students,vector<Info_colegio> &colegios)
{
    for(int x = 0; x < n_colegios; x++){
        aluxcol[x] = colegios[x].num_alu;
        previousAluxCol[x] = colegios[x].num_alu;
        bestAluxCol[x] = colegios[x].num_alu;
        aluVulxCol[x] = colegios[x].prioritario;
        previousAluVulxCol[x] = colegios[x].prioritario;
        bestAluVulxCol[x] = colegios[x].prioritario;

    }
    ///////////////////////////////////////////////////
    /// Se crear un arreglo donde el el valor es la posición del estudiante sep
    ///////////////////////////////////////////////////
    for(int x=0; x < n_students; x++) {
        alumnosSep[x] = students[x].sep;
    }
}


double round_n(double x)
{
    double digits = pow(10.0, DECIMAL);
    return trunc(x * digits) / digits;
}


