#include "hip/hip_runtime.h"
#include <kernel.cuh>

__constant__ int d_cupoArray[85];
__constant__ double d_alpha[3];

__global__ void newSolution_kernel(
    double *d_array_current_Solution,
    double *d_array_current_Solution_cup,
    double *d_alpha_current_Solution_seg,
    double *d_alpha_current_Solution_dis,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col,
    const int n_students,
    const int n_colegios,
    const int space_solution,
    const int n_thread,
    const double max_dist,
    const int* __restrict__ d_alumnosSep,
    const int totalVuln,
    const int* __restrict__ d_aluxcol,
    const int* __restrict__ d_aluVulxCol,
    const int* __restrict__ d_currentSolution,
    const double* __restrict__ d_distMat,
    const int* __restrict__ d_shuffle_students,
    const int* __restrict__ d_shuffle_colegios,
    const double* __restrict__ d_currentVars,
    size_t pitch){

    /// Inicializa variables en 0
    int aluchange,
            colchange,
            newSchool,
            aluVulCol= 0,
            aluNoVulCol= 0,
            totalAluCol= 0,
            myID = blockIdx.x * blockDim.x + threadIdx.x,
            currentSchool,
            salto= n_thread,
            myCurrentAluxCol = 0,
            myCurrentAluVulxCol = 0,
            myNewAluxCol = 0,
            myNewAluVulxCol = 0;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            var1,
            var2,
            var3,
            sumDist = 0.0;
    /// Inicializa arrays
    aluchange = d_shuffle_students[myID%n_students]; 
    colchange = d_shuffle_colegios[blockIdx.x%n_colegios];
    currentSchool = d_currentSolution[aluchange];
    newSchool = colchange;
    sumDist= d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];
    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    
    sumDist-=(d_distMat[aluchange * pitch / sizeof(double) + currentSchool]);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual 
    totalcostCupo-=((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela nueva
    totalcostCupo-=((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2)));
    ////////////////////////////////////////////////////////////////
    /////// Realiza Movimiento
    ////////////////////////////////////////////////////////////////
    //ELimina el estudiante de la escuela actual
    myCurrentAluxCol = d_aluxcol[currentSchool]-1;
    myCurrentAluVulxCol = d_aluVulxCol[currentSchool]-d_alumnosSep[aluchange];
    //Asigna al estudiante a la nueva escuela
    myNewAluxCol = d_aluxcol[newSchool]+1;
    myNewAluVulxCol = d_aluVulxCol[newSchool]+d_alumnosSep[aluchange];

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist+=(d_distMat[aluchange * pitch / sizeof(double) + newSchool]);
    // seg de la escuela actual
    totalAluCol = myCurrentAluxCol;
    aluVulCol = myCurrentAluVulxCol;
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual
    totalcostCupo+=((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    
    // seg de la escuela antigua
    totalAluCol = myNewAluxCol;
    aluVulCol = myNewAluVulxCol;
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela antigua
    totalcostCupo+=(((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2))));


    var1 = (sumDist/n_students);
    var1= (var1/max_dist);

    var2 = (totalSesc/2.0);
    //cout << var2 << "\n";
    var3 = (totalcostCupo /n_colegios);
    d_array_current_Solution[myID]= (d_alpha[0]*var1+(d_alpha[1]*var2+(d_alpha[2]*var3+0)));
    d_array_current_Solution_col[myID] = colchange;
    d_array_current_Solution_alu[myID] = aluchange;
    d_array_current_Solution_cup[myID] = totalcostCupo;
    d_alpha_current_Solution_seg[myID] = totalSesc;
    d_alpha_current_Solution_dis[myID] = sumDist;

    
}

__global__ void reduce_block_kernel(
    double *d_array_current_Solution,
    double *d_array_current_Solution_cup,
    double *d_alpha_current_Solution_seg,
    double *d_alpha_current_Solution_dis,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col,
    int space_solution,
    int* d_alumnosSep,
    int* d_aluxcol,
    int* d_aluVulxCol,
    int* d_currentSolution,
    double *d_currentVars,
    double *d_costCurrentSolution
    ){

    int myID = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("%d %d %d\n",myID,space_solution,blockDim.x);
    int salto= space_solution;
    while(salto){
        if(salto-(myID+1)>myID){
            if(d_array_current_Solution[salto-(myID+1)]<d_array_current_Solution[myID]){
                d_array_current_Solution[myID]=d_array_current_Solution[salto-(myID+1)];
                d_array_current_Solution_cup[myID]=d_array_current_Solution_cup[salto-(myID+1)];
                d_alpha_current_Solution_seg[myID]=d_alpha_current_Solution_seg[salto-(myID+1)];
                d_alpha_current_Solution_dis[myID]=d_alpha_current_Solution_dis[salto-(myID+1)];
                d_array_current_Solution_alu[myID]=d_array_current_Solution_alu[salto-(myID+1)];
                d_array_current_Solution_col[myID]=d_array_current_Solution_col[salto-(myID+1)];
            }
        }
        salto = (salto/2)+(salto&(2-1));
        if(salto==1){
            salto = 0;
        }
        __syncthreads();
    }
    if(myID == 0){
        d_costCurrentSolution[0] = d_array_current_Solution[myID];


        d_currentVars[0] = d_alpha_current_Solution_dis[myID];
        d_currentVars[1] = d_alpha_current_Solution_seg[myID];
        d_currentVars[2] = d_array_current_Solution_cup[myID];

        int aluchange = d_array_current_Solution_alu[0];
        int newSchool = d_array_current_Solution_col[0];
        int currentSchool = d_currentSolution[aluchange];

        d_aluxcol[currentSchool]-=1;
        d_aluVulxCol[currentSchool]-=d_alumnosSep[aluchange];
        
        //Asigna al estudiante a la nueva escuela
        d_currentSolution[aluchange] = newSchool;
        d_aluxcol[newSchool]+=1;
        d_aluVulxCol[newSchool]+=d_alumnosSep[aluchange];
    }
    //printf("%lf\n",d_array_current_Solution[0]);
    
}

__global__ void calculateSolution(
    double *d_array_current_Solution,
    int *d_array_current_Solution_alu,
    int *d_array_current_Solution_col,
    const int n_students,
    const int n_colegios,
    const int n_thread,
    const double max_dist,
    const int* __restrict__ d_alumnosSep,
    int totalVuln,
    int* d_aluxcol,
    int* d_aluVulxCol,
    int* d_currentSolution,
    const double* __restrict__ d_distMat,
    size_t pitch,
    double *d_currentVars,
    double *d_costCurrentSolution){

    int aluchange,
    colchange,
    newSchool,
    aluVulCol= 0,
    aluNoVulCol= 0,
    totalAluCol= 0,
    currentSchool;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            var1,
            var2,
            var3,
            sumDist = 0.0;
    /// Inicializa arrays

    aluchange = d_array_current_Solution_alu[0];
    colchange = d_array_current_Solution_col[0];
    
    currentSchool = d_currentSolution[aluchange];
    //printf("%d,%d, %d\n",aluchange,currentSchool,colchange);
    //printf("%d \t %.20lf | %d %d %d \n",blockIdx.x,d_array_current_Solution[0],d_array_current_Solution_alu[0],d_array_current_Solution_col[0],currentSchool);
    newSchool = colchange;

    
    sumDist= d_currentVars[0];
    totalSesc = d_currentVars[1];
    totalcostCupo = d_currentVars[2];
    //printf("%lf |%lf |%lf |%lf |%d |%d \n",sumDist,totalSesc,totalcostCupo,d_array_current_Solution[0],aluchange,colchange);

    ////////////////////////////////////////////////////////////////
    /////// Descuenta antes de mover
    ////////////////////////////////////////////////////////////////
    // Distancia
    sumDist-=(d_distMat[aluchange * pitch / sizeof(double) + currentSchool]);
    //printf("%lf \n",sumDist);
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    
    //cout << "Alumnos actual escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc-=(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual 
    //printf("%lf \n",totalSesc);
    totalcostCupo-=((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    //printf("%lf \n",totalcostCupo);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    // seg de la escuela nueva
    totalAluCol = d_aluxcol[newSchool];
    //cout << "Alumnos nueva escuela "<< totalAluCol << " " << endl;
    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    
    totalSesc-=(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela nueva

    totalcostCupo-=((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2)));
    //printf("a%d \n",newSchool);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    ////////////////////////////////////////////////////////////////
    /////// Realiza Movimiento
    ////////////////////////////////////////////////////////////////
    //ELimina el estudiante de la escuela actual
    d_aluxcol[currentSchool]-=1;
    d_aluVulxCol[currentSchool]-=d_alumnosSep[aluchange];
    //Asigna al estudiante a la nueva escuela
    d_currentSolution[aluchange] = newSchool;
    d_aluxcol[newSchool]+=1;
    d_aluVulxCol[newSchool]+=d_alumnosSep[aluchange];

    ////////////////////////////////////////////////////////////////
    ////// Calculó despues de mover
    //////////////////////////////////////////////////////////////
    sumDist+=(d_distMat[aluchange * pitch / sizeof(double) + newSchool]);
    
    // seg de la escuela actual
    totalAluCol = d_aluxcol[currentSchool];
    aluVulCol = d_aluVulxCol[currentSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));
    // costcupo escuela actual

    totalcostCupo+=((double)totalAluCol*fabs(((double)d_cupoArray[currentSchool]-totalAluCol)/pow(((double)d_cupoArray[currentSchool]/2),2)));
    //printf("%lf \n",totalcostCupo);
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    // seg de la escuela antigua
    totalAluCol = d_aluxcol[newSchool];

    aluVulCol = d_aluVulxCol[newSchool];
    aluNoVulCol =totalAluCol - aluVulCol;
    totalSesc+=(fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln))));

    // costcupo escuela antigua

    totalcostCupo+=(((double)totalAluCol*fabs(((double)d_cupoArray[newSchool]-totalAluCol)/pow(((double)d_cupoArray[newSchool]/2),2))));
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    d_currentVars[0] = sumDist;
    d_currentVars[1] = totalSesc;
    d_currentVars[2] = totalcostCupo;
    //printf("%lf %lf %lf %d %d %d\n",sumDist,totalSesc,totalcostCupo,aluchange,colchange,currentSchool);
    var1 = (sumDist/n_students);
    var1= (var1/max_dist);
    //cout << var1 << "\n";
    var2 = (totalSesc/2.0);
    //cout << var2 << "\n";
    var3 = (totalcostCupo /n_colegios);
    d_costCurrentSolution[0] =  (double)((d_alpha[0]*var1)+(d_alpha[1]*var2)+(d_alpha[2]*var3));
    //printf("%lf\n",d_costCurrentSolution[0]);
    d_array_current_Solution[0] = d_costCurrentSolution[0];
    if(d_array_current_Solution[0]!=d_costCurrentSolution[0]){
        printf("ERRORRRRRRRRRRRR no son iguales!!!!!!!!!!!!!!!!!!\n");
        printf("%.10lf\n",d_array_current_Solution[0]);
        printf("%.10lf\n",d_costCurrentSolution[0]);

    }
    //d_array_current_Solution[0] = d_costCurrentSolution[0];
}


__global__ void copyMemSolution(
    int *solution,
    int *new_solution,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        solution[i] = new_solution[i];
    }
}
__global__ void copyMemCol(
    int *col,
    int *new_col,
    int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        col[i] = new_col[i];
    }
}
__global__ void copyVars(
    double *var,
    double *new_var){

    var[threadIdx.x] = new_var[threadIdx.x];
}

__global__ void copyCost(
    double *costCurrentSolution,
    double *new_costCurrentSolution
    ){

        costCurrentSolution[0] = new_costCurrentSolution[0];

    }

inline __device__ double cu_round_n(double x)
{
    double digits = pow(10.0, 16);
    return trunc(x * digits) / digits;
}
